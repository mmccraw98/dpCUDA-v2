#include "hip/hip_runtime.h"
#include "../../include/constants.h"
#include "../../include/functors.h"
#include "../../include/particle/particle.h"
#include "../../include/kernels/kernels.cuh"
#include "../../include/particle/config.h"
#include <iostream>
#include <string>
#include <vector>
#include <cmath>
#include <random>
#include <time.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/fill.h>
#include <thrust/copy.h>
#include <thrust/transform.h>
#include <thrust/transform_reduce.h>
#include <thrust/inner_product.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/permutation_iterator.h>
#include <thrust/functional.h>

#define CUDA_CHECK(call)                                                    \
    {                                                                       \
        hipError_t err = call;                                             \
        if (err != hipSuccess) {                                           \
            std::cerr << "CUDA error in " << __FILE__ << " at line "        \
                      << __LINE__ << ": " << hipGetErrorString(err) << "\n"; \
            std::exit(EXIT_FAILURE);                                        \
        }                                                                   \
    }

Particle::Particle() {
}

Particle::~Particle() {
    clearDynamicVariables();
    clearGeometricVariables();
}

void Particle::initializeFromConfig(const BaseParticleConfig& config) {
}

// ----------------------------------------------------------------------
// ----------------------- Template Methods -----------------------------
// ----------------------------------------------------------------------

std::unordered_map<std::string, std::any> Particle::getArrayMap() {
    std::unordered_map<std::string, std::any> array_map;
    array_map["d_positions_x"]          = &d_positions_x;
    array_map["d_positions_y"]          = &d_positions_y;
    array_map["d_last_neigh_positions_x"]     = &d_last_neigh_positions_x;
    array_map["d_last_neigh_positions_y"]     = &d_last_neigh_positions_y;
    array_map["d_last_cell_positions_x"]     = &d_last_cell_positions_x;
    array_map["d_last_cell_positions_y"]     = &d_last_cell_positions_y;
    array_map["d_neigh_displacements_sq"]      = &d_neigh_displacements_sq;
    array_map["d_cell_displacements_sq"]      = &d_cell_displacements_sq;
    array_map["d_velocities_x"]         = &d_velocities_x;
    array_map["d_velocities_y"]         = &d_velocities_y;
    array_map["d_forces_x"]             = &d_forces_x;
    array_map["d_forces_y"]             = &d_forces_y;
    array_map["d_radii"]              = &d_radii;
    array_map["d_masses"]             = &d_masses;
    array_map["d_potential_energy"]   = &d_potential_energy;
    array_map["d_kinetic_energy"]     = &d_kinetic_energy;
    array_map["d_neighbor_list"]      = &d_neighbor_list;
    array_map["d_num_neighbors"]      = &d_num_neighbors;
    array_map["d_cell_index"]         = &d_cell_index;
    array_map["d_sorted_cell_index"]  = &d_sorted_cell_index;
    array_map["d_particle_index"]     = &d_particle_index;
    array_map["d_cell_start"]         = &d_cell_start;
    return array_map;
}

std::string Particle::getArrayType(const std::string& array_name) {
    std::unordered_map<std::string, std::string> array_type_map;
    array_type_map["d_positions_x"]          = "double";
    array_type_map["d_positions_y"]          = "double";
    array_type_map["d_last_neigh_positions_x"]     = "double";
    array_type_map["d_last_neigh_positions_y"]     = "double";
    array_type_map["d_last_cell_positions_x"]     = "double";
    array_type_map["d_last_cell_positions_y"]     = "double";
    array_type_map["d_neigh_displacements_sq"]      = "double";
    array_type_map["d_cell_displacements_sq"]      = "double";
    array_type_map["d_velocities_x"]         = "double";
    array_type_map["d_velocities_y"]         = "double";
    array_type_map["d_forces_x"]             = "double";
    array_type_map["d_forces_y"]             = "double";
    array_type_map["d_radii"]              = "double";
    array_type_map["d_masses"]             = "double";
    array_type_map["d_potential_energy"]   = "double";
    array_type_map["d_kinetic_energy"]     = "double";
    array_type_map["d_box_size"]           = "double";
    array_type_map["d_neighbor_list"]      = "long";
    array_type_map["d_num_neighbors"]      = "long";
    array_type_map["d_cell_index"]         = "long";
    array_type_map["d_sorted_cell_index"]  = "long";
    array_type_map["d_particle_index"]     = "long";
    array_type_map["d_cell_start"]         = "long";
    return array_type_map[array_name];
}

// ----------------------------------------------------------------------
// -------------------- Universally Defined Methods ---------------------
// ----------------------------------------------------------------------

void Particle::setNeighborListUpdateMethod(std::string method_name) {
    if (method_name == "cell") {
        std::cout << "Particle::setNeighborListUpdateMethod: Setting neighbor list update method to cell" << std::endl;
        this->updateNeighborListPtr = &Particle::updateCellNeighborList;
        this->checkForNeighborUpdatePtr = &Particle::checkForCellUpdate;
    } else if (method_name == "verlet") {
        std::cout << "Particle::setNeighborListUpdateMethod: Setting neighbor list update method to verlet" << std::endl;
        this->updateNeighborListPtr = &Particle::updateNeighborList;
        this->checkForNeighborUpdatePtr = &Particle::checkForNeighborUpdate;
    } else if (method_name == "none") {
        std::cout << "Particle::setNeighborListUpdateMethod: Setting neighbor list update method to none" << std::endl;
        throw std::invalid_argument("Particle::setNeighborListUpdateMethod: 'none' neighbor list update method not implemented: " + method_name);
    } else {
        throw std::invalid_argument("Particle::setNeighborListUpdateMethod: Invalid method name: " + method_name);
    }
}

void Particle::setSeed(long seed) {
    if (seed == -1) {
        seed = time(0);
    }
    this->seed = seed;
    srand(seed);
}

void Particle::setNumParticles(long n_particles) {
    this->n_particles = n_particles;
    syncNumParticles();
}

void Particle::syncNumParticles() {
    hipError_t cuda_err = hipMemcpyToSymbol(HIP_SYMBOL(d_n_particles), &n_particles, sizeof(long));
    if (cuda_err != hipSuccess) {
        std::cerr << "Particle::syncNumParticles: Error copying n_particles to device: " << hipGetErrorString(cuda_err) << std::endl;
        exit(EXIT_FAILURE);
    }
}

void Particle::setDegreesOfFreedom() {
    this->n_dof = n_particles * N_DIM;
}

void Particle::setNumVertices(long n_vertices) {
    this->n_vertices = n_vertices;
    syncNumVertices();
}

void Particle::syncNumVertices() {
    hipError_t cuda_err = hipMemcpyToSymbol(HIP_SYMBOL(d_n_vertices), &n_vertices, sizeof(long));
    if (cuda_err != hipSuccess) {
        std::cerr << "Particle::syncNumVertices: Error copying n_vertices to device: " << hipGetErrorString(cuda_err) << std::endl;
        exit(EXIT_FAILURE);
    }
}

void Particle::setParticleCounts(long n_particles, long n_vertices) {
    setNumParticles(n_particles);
    setNumVertices(n_vertices);
    setDegreesOfFreedom();
    initDynamicVariables();
    initGeometricVariables();
}

void Particle::setKernelDimensions(long dim_block) {
    int maxThreadsPerBlock;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    maxThreadsPerBlock = deviceProp.maxThreadsPerBlock;
    std::cout << "CUDA Info: Particle::setKernelDimensions: Max threads per block: " << maxThreadsPerBlock << std::endl;
    if (dim_block > maxThreadsPerBlock) {
        std::cout << "WARNING: Particle::setKernelDimensions: dim_block exceeds maxThreadsPerBlock, adjusting to maxThreadsPerBlock" << std::endl;
        dim_block = maxThreadsPerBlock;
    }
    this->dim_block = dim_block;
    // Implement some particle-specific logic to define the grid dimensions
    // Then, sync
    std::cout << "WARNING: Particle::setKernelDimensions: Not Implemented" << std::endl;
    syncKernelDimensions();
}

void Particle::syncKernelDimensions() {
    hipError_t cuda_err;
    cuda_err = hipMemcpyToSymbol(HIP_SYMBOL(d_dim_block), &dim_block, sizeof(long));
    if (cuda_err != hipSuccess) {
        std::cerr << "Particle::syncKernelDimensions: Error copying dim_block to device: " << hipGetErrorString(cuda_err) << std::endl;
        exit(EXIT_FAILURE);
    }
    cuda_err = hipMemcpyToSymbol(HIP_SYMBOL(d_dim_grid), &dim_grid, sizeof(long));
    if (cuda_err != hipSuccess) {
        std::cerr << "Particle::syncKernelDimensions: Error copying dim_grid to device: " << hipGetErrorString(cuda_err) << std::endl;
        exit(EXIT_FAILURE);
    }
    cuda_err = hipMemcpyToSymbol(HIP_SYMBOL(d_dim_vertex_grid), &dim_vertex_grid, sizeof(long));
    if (cuda_err != hipSuccess) {
        std::cerr << "Particle::syncKernelDimensions: Error copying dim_vertex_grid to device: " << hipGetErrorString(cuda_err) << std::endl;
        exit(EXIT_FAILURE);
    }
}

void Particle::initDynamicVariables() {
    // Resize the device vectors
    d_positions_x.resize(n_particles);
    d_positions_y.resize(n_particles);
    d_last_neigh_positions_x.resize(n_particles);
    d_last_neigh_positions_y.resize(n_particles);
    d_last_cell_positions_x.resize(n_particles);
    d_last_cell_positions_y.resize(n_particles);
    d_neigh_displacements_sq.resize(n_particles);
    d_cell_displacements_sq.resize(n_particles);
    d_velocities_x.resize(n_particles);
    d_velocities_y.resize(n_particles);
    d_forces_x.resize(n_particles);
    d_forces_y.resize(n_particles);
    d_radii.resize(n_particles);
    d_masses.resize(n_particles);
    d_potential_energy.resize(n_particles);
    d_kinetic_energy.resize(n_particles);
    d_neighbor_list.resize(n_particles);
    d_num_neighbors.resize(n_particles);
    d_temp_positions_x.resize(n_particles);
    d_temp_positions_y.resize(n_particles);
    d_temp_forces_x.resize(n_particles);
    d_temp_forces_y.resize(n_particles);
    d_temp_velocities_x.resize(n_particles);
    d_temp_velocities_y.resize(n_particles);
    d_temp_masses.resize(n_particles);
    d_temp_radii.resize(n_particles);

    thrust::fill(d_positions_x.begin(), d_positions_x.end(), 0.0);
    thrust::fill(d_positions_y.begin(), d_positions_y.end(), 0.0);
    thrust::fill(d_last_neigh_positions_x.begin(), d_last_neigh_positions_x.end(), 0.0);
    thrust::fill(d_last_neigh_positions_y.begin(), d_last_neigh_positions_y.end(), 0.0);
    thrust::fill(d_last_cell_positions_x.begin(), d_last_cell_positions_x.end(), 0.0);
    thrust::fill(d_last_cell_positions_y.begin(), d_last_cell_positions_y.end(), 0.0);
    thrust::fill(d_neigh_displacements_sq.begin(), d_neigh_displacements_sq.end(), 0.0);
    thrust::fill(d_cell_displacements_sq.begin(), d_cell_displacements_sq.end(), 0.0);
    thrust::fill(d_velocities_x.begin(), d_velocities_x.end(), 0.0);
    thrust::fill(d_velocities_y.begin(), d_velocities_y.end(), 0.0);
    thrust::fill(d_forces_x.begin(), d_forces_x.end(), 0.0);
    thrust::fill(d_forces_y.begin(), d_forces_y.end(), 0.0);
    thrust::fill(d_radii.begin(), d_radii.end(), 0.0);
    thrust::fill(d_masses.begin(), d_masses.end(), 0.0);
    thrust::fill(d_potential_energy.begin(), d_potential_energy.end(), 0.0);
    thrust::fill(d_kinetic_energy.begin(), d_kinetic_energy.end(), 0.0);
    thrust::fill(d_temp_positions_x.begin(), d_temp_positions_x.end(), 0.0);
    thrust::fill(d_temp_positions_y.begin(), d_temp_positions_y.end(), 0.0);
    thrust::fill(d_temp_forces_x.begin(), d_temp_forces_x.end(), 0.0);
    thrust::fill(d_temp_forces_y.begin(), d_temp_forces_y.end(), 0.0);
    thrust::fill(d_temp_velocities_x.begin(), d_temp_velocities_x.end(), 0.0);
    thrust::fill(d_temp_velocities_y.begin(), d_temp_velocities_y.end(), 0.0);
    thrust::fill(d_temp_masses.begin(), d_temp_masses.end(), 0.0);
    thrust::fill(d_temp_radii.begin(), d_temp_radii.end(), 0.0);

    // max_neighbors = 0;
    // max_neighbors_allocated = 0;
    thrust::fill(d_neighbor_list.begin(), d_neighbor_list.end(), -1L);
    thrust::fill(d_num_neighbors.begin(), d_num_neighbors.end(), max_neighbors);


    // Cast the raw pointers
    d_positions_x_ptr = thrust::raw_pointer_cast(&d_positions_x[0]);
    d_positions_y_ptr = thrust::raw_pointer_cast(&d_positions_y[0]);
    d_last_neigh_positions_x_ptr = thrust::raw_pointer_cast(&d_last_neigh_positions_x[0]);
    d_last_neigh_positions_y_ptr = thrust::raw_pointer_cast(&d_last_neigh_positions_y[0]);
    d_last_cell_positions_x_ptr = thrust::raw_pointer_cast(&d_last_cell_positions_x[0]);
    d_last_cell_positions_y_ptr = thrust::raw_pointer_cast(&d_last_cell_positions_y[0]);
    d_neigh_displacements_sq_ptr = thrust::raw_pointer_cast(&d_neigh_displacements_sq[0]);
    d_cell_displacements_sq_ptr = thrust::raw_pointer_cast(&d_cell_displacements_sq[0]);
    d_velocities_x_ptr = thrust::raw_pointer_cast(&d_velocities_x[0]);
    d_velocities_y_ptr = thrust::raw_pointer_cast(&d_velocities_y[0]);
    d_forces_x_ptr = thrust::raw_pointer_cast(&d_forces_x[0]);
    d_forces_y_ptr = thrust::raw_pointer_cast(&d_forces_y[0]);
    d_radii_ptr = thrust::raw_pointer_cast(&d_radii[0]);
    d_masses_ptr = thrust::raw_pointer_cast(&d_masses[0]);
    d_potential_energy_ptr = thrust::raw_pointer_cast(&d_potential_energy[0]);
    d_kinetic_energy_ptr = thrust::raw_pointer_cast(&d_kinetic_energy[0]);
    d_temp_positions_x_ptr = thrust::raw_pointer_cast(&d_temp_positions_x[0]);
    d_temp_positions_y_ptr = thrust::raw_pointer_cast(&d_temp_positions_y[0]);
    d_temp_forces_x_ptr = thrust::raw_pointer_cast(&d_temp_forces_x[0]);
    d_temp_forces_y_ptr = thrust::raw_pointer_cast(&d_temp_forces_y[0]);
    d_temp_velocities_x_ptr = thrust::raw_pointer_cast(&d_temp_velocities_x[0]);
    d_temp_velocities_y_ptr = thrust::raw_pointer_cast(&d_temp_velocities_y[0]);
    d_temp_masses_ptr = thrust::raw_pointer_cast(&d_temp_masses[0]);
    d_temp_radii_ptr = thrust::raw_pointer_cast(&d_temp_radii[0]);
}

void Particle::clearDynamicVariables() {
    // Clear the device vectors
    d_positions_x.clear();
    d_positions_y.clear();
    d_last_neigh_positions_x.clear();
    d_last_neigh_positions_y.clear();
    d_neigh_displacements_sq.clear();
    d_cell_displacements_sq.clear();
    d_last_cell_positions_x.clear();
    d_last_cell_positions_y.clear();
    d_velocities_x.clear();
    d_velocities_y.clear();
    d_forces_x.clear();
    d_forces_y.clear();
    d_radii.clear();
    d_masses.clear();
    d_potential_energy.clear();
    d_kinetic_energy.clear();
    d_neighbor_list.clear();
    d_num_neighbors.clear();
    d_cell_index.clear();
    d_sorted_cell_index.clear();
    d_particle_index.clear();
    d_cell_start.clear();
    d_temp_positions_x.clear();
    d_temp_positions_y.clear();
    d_temp_forces_x.clear();
    d_temp_forces_y.clear();
    d_temp_velocities_x.clear();
    d_temp_velocities_y.clear();
    d_temp_masses.clear();
    d_temp_radii.clear();

    // Clear the pointers
    d_positions_x_ptr = nullptr;
    d_positions_y_ptr = nullptr;
    d_last_neigh_positions_x_ptr = nullptr;
    d_last_neigh_positions_y_ptr = nullptr;
    d_last_cell_positions_x_ptr = nullptr;
    d_last_cell_positions_y_ptr = nullptr;
    d_neigh_displacements_sq_ptr = nullptr;
    d_cell_displacements_sq_ptr = nullptr;
    d_velocities_x_ptr = nullptr;
    d_velocities_y_ptr = nullptr;
    d_forces_x_ptr = nullptr;
    d_forces_y_ptr = nullptr;
    d_radii_ptr = nullptr;
    d_masses_ptr = nullptr;
    d_potential_energy_ptr = nullptr;
    d_kinetic_energy_ptr = nullptr;
    d_cell_index_ptr = nullptr;
    d_sorted_cell_index_ptr = nullptr;
    d_particle_index_ptr = nullptr;
    d_cell_start_ptr = nullptr;
    d_temp_positions_x_ptr = nullptr;
    d_temp_positions_y_ptr = nullptr;
    d_temp_forces_x_ptr = nullptr;
    d_temp_forces_y_ptr = nullptr;
    d_temp_velocities_x_ptr = nullptr;
    d_temp_velocities_y_ptr = nullptr;
    d_temp_masses_ptr = nullptr;
    d_temp_radii_ptr = nullptr;
}

void Particle::setBoxSize(const thrust::host_vector<double>& box_size) {
    if (box_size.size() != N_DIM) {
        throw std::invalid_argument("Particle::setBoxSize: Error box_size (" + std::to_string(box_size.size()) + ")" + " != " + std::to_string(N_DIM) + " elements");
    }
    hipError_t cuda_err = hipMemcpyToSymbol(HIP_SYMBOL(d_box_size), box_size.data(), sizeof(double) * N_DIM);
    if (cuda_err != hipSuccess) {
        std::cerr << "Particle::setBoxSize: Error copying box size to device: " << hipGetErrorString(cuda_err) << std::endl;
        exit(EXIT_FAILURE);
    }
}

thrust::host_vector<double> Particle::getBoxSize() {
    thrust::host_vector<double> box_size(N_DIM);
    hipError_t cuda_err = hipMemcpyFromSymbol(&box_size[0], HIP_SYMBOL(d_box_size), sizeof(double) * N_DIM);
    if (cuda_err != hipSuccess) {
        std::cerr << "Particle::getBoxSize: Error copying box size to host: " << hipGetErrorString(cuda_err) << std::endl;
        exit(EXIT_FAILURE);
    }
    return box_size;
}

void Particle::syncNeighborList() {
    hipError_t cuda_err = hipMemcpyToSymbol(HIP_SYMBOL(d_max_neighbors_allocated), &this->max_neighbors_allocated, sizeof(this->max_neighbors_allocated));
    if (cuda_err != hipSuccess) {
        std::cerr << "Particle::syncNeighborList: Error copying max_neighbors_allocated to device: " << hipGetErrorString(cuda_err) << std::endl;
        exit(EXIT_FAILURE);
    }
    long* neighbor_list_ptr = thrust::raw_pointer_cast(&d_neighbor_list[0]);
    cuda_err = hipMemcpyToSymbol(HIP_SYMBOL(d_neighbor_list_ptr), &neighbor_list_ptr, sizeof(neighbor_list_ptr));
    if (cuda_err != hipSuccess) {
        std::cerr << "Particle::syncNeighborList: Error copying d_neighbor_list_ptr to device: " << hipGetErrorString(cuda_err) << std::endl;
        exit(EXIT_FAILURE);
    }
    long* num_neighbors_ptr = thrust::raw_pointer_cast(&d_num_neighbors[0]);
    cuda_err = hipMemcpyToSymbol(HIP_SYMBOL(d_num_neighbors_ptr), &num_neighbors_ptr, sizeof(num_neighbors_ptr));
    if (cuda_err != hipSuccess) {
        std::cerr << "Particle::syncNeighborList: Error copying d_num_neighbors_ptr to device: " << hipGetErrorString(cuda_err) << std::endl;
        exit(EXIT_FAILURE);
    }
}

void Particle::setEnergyScale(double e, std::string which) {
    hipError_t cuda_err;
    if (which == "c") {
        e_c = e;
        cuda_err = hipMemcpyToSymbol(HIP_SYMBOL(d_e_c), &e_c, sizeof(double));
        if (cuda_err != hipSuccess) {
            std::cerr << "Particle::setEnergyScale: Error copying e_c to device: " << hipGetErrorString(cuda_err) << std::endl;
            exit(EXIT_FAILURE);
        }
    } else if (which == "a") {
        e_a = e;
        cuda_err = hipMemcpyToSymbol(HIP_SYMBOL(d_e_a), &e_a, sizeof(double));
        if (cuda_err != hipSuccess) {
            std::cerr << "Particle::setEnergyScale: Error copying e_a to device: " << hipGetErrorString(cuda_err) << std::endl;
            exit(EXIT_FAILURE);
        }
    } else if (which == "b") {
        e_b = e;
        cuda_err = hipMemcpyToSymbol(HIP_SYMBOL(d_e_b), &e_b, sizeof(double));
        if (cuda_err != hipSuccess) {
            std::cerr << "Particle::setEnergyScale: Error copying e_b to device: " << hipGetErrorString(cuda_err) << std::endl;
            exit(EXIT_FAILURE);
        }
    } else if (which == "l") {
        e_l = e;
        cuda_err = hipMemcpyToSymbol(HIP_SYMBOL(d_e_l), &e_l, sizeof(double));
        if (cuda_err != hipSuccess) {
            std::cerr << "Particle::setEnergyScale: Error copying e_l to device: " << hipGetErrorString(cuda_err) << std::endl;
            exit(EXIT_FAILURE);
        }
    } else {
        throw std::invalid_argument("Particle::setEnergyScale: which must be 'c', 'a', 'b', or 'l', not " + which);
    }
}

double Particle::getEnergyScale(std::string which) {
    if (which == "c") {
        return e_c;
    } else if (which == "a") {
        return e_a;
    } else if (which == "b") {
        return e_b;
    } else if (which == "l") {
        return e_l;
    } else {
        throw std::invalid_argument("Particle::getEnergyScale: which must be 'c', 'a', 'b', or 'l', not " + which);
    }
}

void Particle::setAllEnergyScales(double e_c, double e_a, double e_b, double e_l) {
    setEnergyScale(e_c, "c");
    setEnergyScale(e_a, "a");
    setEnergyScale(e_b, "b");
    setEnergyScale(e_l, "l");
}

void Particle::setExponent(double n, std::string which) {
    hipError_t cuda_err;
    if (which == "c") {
        n_c = n;
        cuda_err = hipMemcpyToSymbol(HIP_SYMBOL(d_n_c), &n_c, sizeof(double));
        if (cuda_err != hipSuccess) {
            std::cerr << "Particle::setExponent: Error copying n_c to device: " << hipGetErrorString(cuda_err) << std::endl;
            exit(EXIT_FAILURE);
        }
    } else if (which == "a") {
        n_a = n;
        cuda_err = hipMemcpyToSymbol(HIP_SYMBOL(d_n_a), &n_a, sizeof(double));
        if (cuda_err != hipSuccess) {
            std::cerr << "Particle::setExponent: Error copying n_a to device: " << hipGetErrorString(cuda_err) << std::endl;
            exit(EXIT_FAILURE);
        }
    } else if (which == "b") {
        n_b = n;
        cuda_err = hipMemcpyToSymbol(HIP_SYMBOL(d_n_b), &n_b, sizeof(double));
        if (cuda_err != hipSuccess) {
            std::cerr << "Particle::setExponent: Error copying n_b to device: " << hipGetErrorString(cuda_err) << std::endl;
            exit(EXIT_FAILURE);
        }
    } else if (which == "l") {
        n_l = n;
        cuda_err = hipMemcpyToSymbol(HIP_SYMBOL(d_n_l), &n_l, sizeof(double));
        if (cuda_err != hipSuccess) {
            std::cerr << "Particle::setExponent: Error copying n_l to device: " << hipGetErrorString(cuda_err) << std::endl;
            exit(EXIT_FAILURE);
        }
    } else {
        throw std::invalid_argument("Particle::setExponent: which must be 'c', 'a', 'b', or 'l', not " + which);
    }
}

void Particle::setAllExponents(double n_c, double n_a, double n_b, double n_l) {
    setExponent(n_c, "c");
    setExponent(n_a, "a");
    setExponent(n_b, "b");
    setExponent(n_l, "l");
}

double Particle::getExponent(std::string which) {
    if (which == "c") {
        return n_c;
    } else if (which == "a") {
        return n_a;
    } else if (which == "b") {
        return n_b;
    } else if (which == "l") {
        return n_l;
    } else {
        throw std::invalid_argument("Particle::getExponent: which must be 'c', 'a', 'b', or 'l', not " + which);
    }
}

void Particle::initializeBox(double packing_fraction) {
    // set the box size to an arbitrary initial value
    double side_length = 1.0;
    thrust::host_vector<double> box_size(N_DIM, side_length);
    setBoxSize(box_size);
    // then rescale the box size to the desired packing fraction
    scaleToPackingFraction(packing_fraction);
}

void Particle::setRandomUniform(thrust::device_vector<double>& values, double min, double max) {
    thrust::counting_iterator<long> index_sequence_begin(seed);
    thrust::transform(index_sequence_begin, index_sequence_begin + values.size(), values.begin(), RandomUniform(min, max, seed));
}

void Particle::setRandomNormal(thrust::device_vector<double>& values, double mean, double stddev) {
    std::cout << "Set: This does not work yet" << std::endl;
    thrust::counting_iterator<long> index_sequence_begin(seed);
    thrust::transform(index_sequence_begin, index_sequence_begin + values.size(), values.begin(), RandomNormal(mean, stddev, seed));
}

void Particle::setRandomPositions() {
    thrust::host_vector<double> box_size = getBoxSize();
    setRandomUniform(d_positions_x, 0.0, box_size[0]);
    setRandomUniform(d_positions_y, 0.0, box_size[1]);
}

void Particle::removeMeanVelocities() {
    std::cout << "Remove: This does not work yet" << std::endl;
    // kernelRemoveMeanVelocities<<<1, N_DIM>>>(d_velocities_ptr);
    // hipDeviceSynchronize();
}

void Particle::scaleVelocitiesToTemperature(double temperature) {
    double current_temp = calculateTemperature();
    // TODO: this should be a single kernel
    thrust::transform(d_velocities_x.begin(), d_velocities_x.end(), thrust::make_constant_iterator(std::sqrt(temperature / current_temp)), d_velocities_x.begin(), thrust::multiplies<double>());
    thrust::transform(d_velocities_y.begin(), d_velocities_y.end(), thrust::make_constant_iterator(std::sqrt(temperature / current_temp)), d_velocities_y.begin(), thrust::multiplies<double>());
}

void Particle::setRandomVelocities(double temperature) {
    setRandomNormal(d_velocities_x, 0.0, std::sqrt(temperature));
    setRandomNormal(d_velocities_y, 0.0, std::sqrt(temperature));
    removeMeanVelocities();
    scaleVelocitiesToTemperature(temperature);
    // thrust::fill(d_velocities.begin(), d_velocities.end(), 0.0);
}

double Particle::getDiameter(std::string which) {
    if (which == "min") {
        return 2.0 * *thrust::min_element(d_radii.begin(), d_radii.end());
    } else if (which == "max") {
        return 2.0 * *thrust::max_element(d_radii.begin(), d_radii.end());
    } else if (which == "mean") {
        return 2.0 * thrust::reduce(d_radii.begin(), d_radii.end()) / d_radii.size();
    } else {
        throw std::invalid_argument("Particle::getDiameter: which must be 'min', 'max', or 'mean', not " + which);
    }
}

void Particle::setBiDispersity(double size_ratio, double count_ratio) {
    if (size_ratio < 1.0) {
        throw std::invalid_argument("Particle::setBiDispersity: size_ratio must be > 1.0");
    }
    if (count_ratio < 0.0 || count_ratio > 1.0) {
        throw std::invalid_argument("Particle::setBiDispersity: count_ratio must be < 1.0 and > 0.0");
    }
    thrust::host_vector<double> radii(n_particles);
    long n_large = static_cast<long>(n_particles * count_ratio);
    double diam_large = size_ratio;
    double diam_small = 1.0;
    for (long i = 0; i < n_large; i++) {
        radii[i] = diam_large / 2.0;
    }
    for (long i = n_large; i < n_particles; i++) {
        radii[i] = diam_small / 2.0;
    }
    setArray("d_radii", radii);
}

double Particle::getBoxArea() {
    thrust::host_vector<double> box_size = getBoxSize();
    return thrust::reduce(box_size.begin(), box_size.end(), 1.0, thrust::multiplies<double>());
}

double Particle::getPackingFraction() {
    double box_area = getBoxArea();
    double area = getArea();
    return area / box_area;
}

double Particle::getDensity() {
    return getPackingFraction() - getOverlapFraction();
}

void Particle::scaleToPackingFraction(double packing_fraction) {
    double new_side_length = std::pow(getArea() / packing_fraction, 1.0 / N_DIM);
    double side_length = std::pow(getBoxArea(), 1.0 / N_DIM);
    scalePositions(new_side_length / side_length);
    setBoxSize(thrust::host_vector<double>(N_DIM, new_side_length));
}

double Particle::totalKineticEnergy() const {
    return thrust::reduce(d_kinetic_energy.begin(), d_kinetic_energy.end(), 0.0, thrust::plus<double>());
}

double Particle::totalPotentialEnergy() const {
    return thrust::reduce(d_potential_energy.begin(), d_potential_energy.end(), 0.0, thrust::plus<double>());
}

double Particle::totalEnergy() const {
    return totalKineticEnergy() + totalPotentialEnergy();
}

void Particle::scalePositions(double scale_factor) {
    thrust::transform(d_positions_x.begin(), d_positions_x.end(), thrust::make_constant_iterator(scale_factor), d_positions_x.begin(), thrust::multiplies<double>());
    thrust::transform(d_positions_y.begin(), d_positions_y.end(), thrust::make_constant_iterator(scale_factor), d_positions_y.begin(), thrust::multiplies<double>());
}

void Particle::updatePositions(double dt) {
    if (isnan(d_positions_x[0])) {
        std::cout << "Particle::updatePositions: NaN in positions for particle 0: pos_x=" << d_positions_x[0] << ", pos_y=" << d_positions_y[0] << std::endl;
        exit(EXIT_SUCCESS);
    }
    kernelUpdatePositions<<<dim_grid, dim_block>>>(d_positions_x_ptr, d_positions_y_ptr, d_last_neigh_positions_x_ptr, d_last_neigh_positions_y_ptr, d_last_cell_positions_x_ptr, d_last_cell_positions_y_ptr, d_neigh_displacements_sq_ptr, d_cell_displacements_sq_ptr, d_velocities_x_ptr, d_velocities_y_ptr, dt);
}

void Particle::updateVelocities(double dt) {
    kernelUpdateVelocities<<<dim_grid, dim_block>>>(d_velocities_x_ptr, d_velocities_y_ptr, d_forces_x_ptr, d_forces_y_ptr, d_masses_ptr, dt);
}

double Particle::getMaxSquaredNeighborDisplacement() {
    return thrust::reduce(d_neigh_displacements_sq.begin(), d_neigh_displacements_sq.end(), 0.0, thrust::maximum<double>());
}

double Particle::getMaxSquaredCellDisplacement() {
    return thrust::reduce(d_cell_displacements_sq.begin(), d_cell_displacements_sq.end(), 0.0, thrust::maximum<double>());
}

void Particle::updateNeighborList() {
    thrust::fill(d_neighbor_list.begin(), d_neighbor_list.end(), -1L);
    kernelUpdateNeighborList<<<dim_grid, dim_block>>>(d_positions_x_ptr, d_positions_y_ptr, d_last_neigh_positions_x_ptr, d_last_neigh_positions_y_ptr, d_neigh_displacements_sq_ptr, neighbor_cutoff);
    max_neighbors = thrust::reduce(d_num_neighbors.begin(), d_num_neighbors.end(), -1L, thrust::maximum<long>());
    if (max_neighbors > max_neighbors_allocated) {
        max_neighbors_allocated = std::pow(2, std::ceil(std::log2(max_neighbors)));
        std::cout << "Particle::updateNeighborList: Resizing neighbor list to " << max_neighbors_allocated << std::endl;
        d_neighbor_list.resize(n_particles * max_neighbors_allocated);
        thrust::fill(d_neighbor_list.begin(), d_neighbor_list.end(), -1L);
        syncNeighborList();
        kernelUpdateNeighborList<<<dim_grid, dim_block>>>(d_positions_x_ptr, d_positions_y_ptr, d_last_neigh_positions_x_ptr, d_last_neigh_positions_y_ptr, d_neigh_displacements_sq_ptr, neighbor_cutoff);
    }
}

void Particle::checkNeighbors() {
    // std::cout << "Particle::checkNeighbors: Checking neighbors" << std::endl;
    (this->*checkForNeighborUpdatePtr)();
}

void Particle::checkForNeighborUpdate() {
    double tolerance = 3.0;
    double max_squared_neighbor_displacement = getMaxSquaredNeighborDisplacement();
    // std::cout << "Particle::checkForNeighborUpdate: Max squared neighbor displacement: " << tolerance * max_squared_neighbor_displacement << " vs " << neighbor_displacement << std::endl;
    if (tolerance * max_squared_neighbor_displacement > neighbor_displacement) {
        // std::cout << "Particle::checkForNeighborUpdate: Updating neighbor list" << std::endl;
        updateNeighborList();
    }
}

void Particle::checkForCellUpdate() {
    double tolerance = 3.0;
    double max_squared_cell_displacement = getMaxSquaredCellDisplacement();
    // std::cout << "Particle::checkForCellUpdate: Max squared cell displacement: " << tolerance * max_squared_cell_displacement << " vs " << cell_displacement << std::endl;
    if (tolerance * max_squared_cell_displacement > cell_displacement) {
        // std::cout << "Particle::checkForCellUpdate: Updating cell list" << std::endl;
        updateCellList();
        updateCellNeighborList();
    } else {
        double max_squared_neighbor_displacement = getMaxSquaredNeighborDisplacement();
        // std::cout << "Particle::checkForCellUpdate: Max squared neighbor displacement: " << tolerance * max_squared_neighbor_displacement << " vs " << neighbor_displacement << std::endl;
        if (tolerance * max_squared_neighbor_displacement > neighbor_displacement) {
            // std::cout << "Particle::checkForNeighborUpdate: Updating neighbor list" << std::endl;
            updateCellNeighborList();
        }
    }
}

void Particle::initializeNeighborList() {
    d_neighbor_list.resize(n_particles * max_neighbors_allocated);
    d_num_neighbors.resize(n_particles);
    thrust::fill(d_num_neighbors.begin(), d_num_neighbors.end(), 0L);
    thrust::fill(d_neighbor_list.begin(), d_neighbor_list.end(), -1L);
    syncNeighborList();
    updateNeighborList();
}

void Particle::setNeighborCutoff(double neighbor_cutoff_multiplier, double neighbor_displacement_multiplier) {
    this->neighbor_cutoff = neighbor_cutoff_multiplier * getDiameter("max");
    this->neighbor_displacement = neighbor_displacement_multiplier * neighbor_cutoff;
    this->max_neighbors_allocated = 4;

    thrust::host_vector<double> box_size = getBoxSize();
    std::cout << "Particle::setNeighborCutoff: Neighbor cutoff set to " << neighbor_cutoff << " and neighbor displacement set to " << neighbor_displacement << " box length: " << box_size[0] << std::endl;
}

void Particle::printNeighborList() {
    thrust::host_vector<long> neighbor_list = getArray<long>("d_neighbor_list");
    thrust::host_vector<long> num_neighbors = getArray<long>("d_num_neighbors");
    for (long i = 0; i < n_particles; i++) {
        std::cout << "Particle " << i << " has " << num_neighbors[i] << " neighbors." << std::endl;
        for (long j = 0; j < num_neighbors[i]; j++) {
            std::cout << "\t\tNeighbor " << j << " of particle " << i << " is " << neighbor_list[i * max_neighbors + j] << std::endl;
        }
    }
}

void Particle::setCellSize(double cell_size_multiplier, double cell_displacement_multiplier) {
    long min_num_cells_dim = 4;  // if there are fewer than 4 cells in one axis, the cell list probably wont work
    double trial_cell_size = cell_size_multiplier * getDiameter("max");
    thrust::host_vector<double> box_size = getBoxSize();
    n_cells_dim = static_cast<long>(std::floor(box_size[0] / trial_cell_size));
    n_cells = n_cells_dim * n_cells_dim;
    if (n_cells_dim < min_num_cells_dim) {
        throw std::runtime_error("Particle::setCellSize: fewer than " + std::to_string(min_num_cells_dim) + " cells in one dimension");
    }
    cell_size = box_size[0] / n_cells_dim;
    cell_displacement = cell_displacement_multiplier * cell_size;
    std::cout << "Particle::setCellSize: Cell size set to " << cell_size << " and cell displacement set to " << cell_displacement << std::endl;
    syncCellList();
}

void Particle::initializeCellList() {
    d_cell_index.resize(n_particles);
    d_sorted_cell_index.resize(n_particles);
    d_particle_index.resize(n_particles);
    d_cell_start.resize(n_cells + 1);

    // TODO: this should be a single kernel
    thrust::fill(d_cell_index.begin(), d_cell_index.end(), -1L);
    thrust::fill(d_sorted_cell_index.begin(), d_sorted_cell_index.end(), -1L);
    thrust::fill(d_particle_index.begin(), d_particle_index.end(), -1L);
    thrust::fill(d_cell_start.begin(), d_cell_start.end(), -1L);

    d_cell_index_ptr = thrust::raw_pointer_cast(d_cell_index.data());
    d_sorted_cell_index_ptr = thrust::raw_pointer_cast(d_sorted_cell_index.data());
    d_particle_index_ptr = thrust::raw_pointer_cast(d_particle_index.data());
    d_cell_start_ptr = thrust::raw_pointer_cast(d_cell_start.data());

    updateCellList();
}

void Particle::syncCellList() {
    hipError_t cuda_err = hipMemcpyToSymbol(HIP_SYMBOL(d_n_cells), &n_cells, sizeof(long));
    if (cuda_err != hipSuccess) {
        std::cerr << "Particle::syncCellList: Error copying n_cells to device: " << hipGetErrorString(cuda_err) << std::endl;
        exit(EXIT_FAILURE);
    }
    cuda_err = hipMemcpyToSymbol(HIP_SYMBOL(d_n_cells_dim), &n_cells_dim, sizeof(long));
    if (cuda_err != hipSuccess) {
        std::cerr << "Particle::syncCellList: Error copying n_cells_dim to device: " << hipGetErrorString(cuda_err) << std::endl;
        exit(EXIT_FAILURE);
    }
    cuda_err = hipMemcpyToSymbol(HIP_SYMBOL(d_cell_size), &cell_size, sizeof(double));
    if (cuda_err != hipSuccess) {
        std::cerr << "Particle::syncCellList: Error copying cell_size to device: " << hipGetErrorString(cuda_err) << std::endl;
        exit(EXIT_FAILURE);
    }
}

void Particle::reorderParticleData() {
    // std::cout << "\tParticle::reorderParticleData: Reordering particle data" << std::endl;
    // do the initial sorting with thrust for convenience
    thrust::sort_by_key(d_sorted_cell_index.begin(), d_sorted_cell_index.end(), d_particle_index.begin());
    // reorder the data by copying into temporary arrays
    // dont bother reordering the displacements, last positions, or energies
    // now print something to see if the ptrs are correct
    kernelReorderParticleData<<<dim_grid, dim_block>>>(d_particle_index_ptr, d_positions_x_ptr, d_positions_y_ptr, d_forces_x_ptr, d_forces_y_ptr, d_velocities_x_ptr, d_velocities_y_ptr, d_masses_ptr, d_radii_ptr, d_temp_positions_x_ptr, d_temp_positions_y_ptr, d_temp_forces_x_ptr, d_temp_forces_y_ptr, d_temp_velocities_x_ptr, d_temp_velocities_y_ptr, d_temp_masses_ptr, d_temp_radii_ptr, d_last_cell_positions_x_ptr, d_last_cell_positions_y_ptr, d_cell_displacements_sq_ptr);

    // swap the pointers
    thrust::swap(d_positions_x_ptr, d_temp_positions_x_ptr);
    thrust::swap(d_positions_y_ptr, d_temp_positions_y_ptr);
    thrust::swap(d_forces_x_ptr, d_temp_forces_x_ptr);
    thrust::swap(d_forces_y_ptr, d_temp_forces_y_ptr);
    thrust::swap(d_velocities_x_ptr, d_temp_velocities_x_ptr);
    thrust::swap(d_velocities_y_ptr, d_temp_velocities_y_ptr);
    thrust::swap(d_masses_ptr, d_temp_masses_ptr);
    thrust::swap(d_radii_ptr, d_temp_radii_ptr);

    assert(d_positions_x_ptr != d_temp_positions_x_ptr);
    assert(d_positions_y_ptr != d_temp_positions_y_ptr);
    assert(d_forces_x_ptr != d_temp_forces_x_ptr);
    assert(d_forces_y_ptr != d_temp_forces_y_ptr);
    assert(d_velocities_x_ptr != d_temp_velocities_x_ptr);
    assert(d_velocities_y_ptr != d_temp_velocities_y_ptr);
    assert(d_masses_ptr != d_temp_masses_ptr);
    assert(d_radii_ptr != d_temp_radii_ptr);
}

void Particle::updateCellList() {
    d_cell_start[n_cells] = n_particles;
    kernelGetCellIndexForParticle<<<dim_grid, dim_block>>>(d_positions_x_ptr, d_positions_y_ptr, d_cell_index_ptr, d_sorted_cell_index_ptr, d_particle_index_ptr);

    reorderParticleData();

    // TODO: this is a kernel over cells - could probably be parallelized better
    long width_offset = 2;
    long width = n_particles / n_cells;
    kernelGetFirstParticleIndexForCell<<<dim_grid, dim_block>>>(d_sorted_cell_index_ptr, d_cell_start_ptr, width_offset, width);
}

// TODO: look into better ways to structure the grid and block sizes
void Particle::updateCellNeighborList() {
    thrust::fill(d_neighbor_list.begin(), d_neighbor_list.end(), -1L);
    kernelUpdateCellNeighborList<<<dim_grid, dim_block>>>(d_positions_x_ptr, d_positions_y_ptr, d_last_neigh_positions_x_ptr, d_last_neigh_positions_y_ptr, neighbor_cutoff, d_cell_index_ptr, d_particle_index_ptr, d_cell_start_ptr, d_neigh_displacements_sq_ptr);
    max_neighbors = thrust::reduce(d_num_neighbors.begin(), d_num_neighbors.end(), -1L, thrust::maximum<long>());
    if (max_neighbors > max_neighbors_allocated) {
        max_neighbors_allocated = std::pow(2, std::ceil(std::log2(max_neighbors)));
        std::cout << "Particle::updateCellNeighborList: Resizing neighbor list to " << max_neighbors_allocated << std::endl;
        d_neighbor_list.resize(n_particles * max_neighbors_allocated);
        thrust::fill(d_neighbor_list.begin(), d_neighbor_list.end(), -1L);
        syncNeighborList();
        kernelUpdateCellNeighborList<<<dim_grid, dim_block>>>(d_positions_x_ptr, d_positions_y_ptr, d_last_neigh_positions_x_ptr, d_last_neigh_positions_y_ptr, neighbor_cutoff, d_cell_index_ptr, d_particle_index_ptr, d_cell_start_ptr, d_neigh_displacements_sq_ptr);
    }
}

// TODO: this should be a single kernel
void Particle::zeroForceAndPotentialEnergy() {
    kernelZeroForceAndPotentialEnergy<<<dim_grid, dim_block>>>(d_forces_x_ptr, d_forces_y_ptr, d_potential_energy_ptr);
}

double Particle::calculateTemperature() {
    calculateKineticEnergy();
    return totalKineticEnergy() * 2.0 / n_dof;
}

double Particle::getTimeUnit() {
    double average_mass = thrust::reduce(d_masses.begin(), d_masses.end()) / n_particles;
    return getDiameter("min") * std::sqrt(average_mass / getEnergyScale("c"));
}

void Particle::setMass(double mass) {
    thrust::fill(d_masses.begin(), d_masses.end(), mass);
}