#include "hip/hip_runtime.h"
#include "../../include/constants.h"
#include "../../include/functors.h"
#include "../../include/particle/particle.h"
#include "../../include/particle/rigid_bumpy.h"
#include "../../include/kernels/kernels.cuh"
#include <iostream>
#include <string>
#include <vector>
#include <cmath>
#include <random>
#include <time.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/fill.h>
#include <thrust/copy.h>
#include <thrust/transform.h>
#include <thrust/transform_reduce.h>
#include <thrust/inner_product.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/permutation_iterator.h>
#include <thrust/functional.h>

RigidBumpy::RigidBumpy() {
}

RigidBumpy::~RigidBumpy() {
}

// ----------------------------------------------------------------------
// --------------------- Overridden Methods -----------------------------
// ----------------------------------------------------------------------


void RigidBumpy::setKernelDimensions(long particle_dim_block, long vertex_dim_block) {
    int maxThreadsPerBlock;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    maxThreadsPerBlock = deviceProp.maxThreadsPerBlock;
    std::cout << "CUDA Info: Particle::setKernelDimensions: Max threads per block: " << maxThreadsPerBlock << std::endl;
    if (particle_dim_block > maxThreadsPerBlock) {
        std::cout << "WARNING: Particle::setKernelDimensions: particle_dim_block exceeds maxThreadsPerBlock, adjusting to maxThreadsPerBlock" << std::endl;
        particle_dim_block = maxThreadsPerBlock;
    }
    if (n_particles <= 0) {
        std::cout << "ERROR: Disk::setKernelDimensions: n_particles is 0.  Set n_particles before setting kernel dimensions." << std::endl;
        exit(EXIT_FAILURE);
    }
    if (n_particles < particle_dim_block) {
        particle_dim_block = n_particles;
    }
    this->particle_dim_block = particle_dim_block;
    this->particle_dim_grid = (n_particles + particle_dim_block - 1) / particle_dim_block;

    if (vertex_dim_block > maxThreadsPerBlock) {
        std::cout << "WARNING: RigidBumpy::setKernelDimensions: vertex_dim_block exceeds maxThreadsPerBlock, adjusting to maxThreadsPerBlock" << std::endl;
        vertex_dim_block = maxThreadsPerBlock;
    }
    if (n_vertices <= 0) {
        std::cout << "ERROR: RigidBumpy::setKernelDimensions: n_vertices is 0.  Set n_vertices before setting kernel dimensions." << std::endl;
        exit(EXIT_FAILURE);
    }
    if (n_vertices < vertex_dim_block) {
        vertex_dim_block = n_vertices;
    }
    this->vertex_dim_block = vertex_dim_block;
    this->vertex_dim_grid = (n_vertices + vertex_dim_block - 1) / vertex_dim_block;

    syncKernelDimensions();
}

void RigidBumpy::initVertexVariables() {
    vertex_positions.resizeAndFill(n_vertices, 0.0, 0.0);
    vertex_velocities.resizeAndFill(n_vertices, 0.0, 0.0);
    vertex_forces.resizeAndFill(n_vertices, 0.0, 0.0);
    vertex_torques.resizeAndFill(n_vertices, 0.0);
    vertex_particle_index.resizeAndFill(n_vertices, 0);
    vertex_masses.resizeAndFill(n_vertices, 0.0);
    vertex_potential_energy.resizeAndFill(n_vertices, 0.0);
}

void RigidBumpy::initDynamicVariables() {
    Particle::initDynamicVariables();
    angles.resizeAndFill(n_particles, 0.0);
    angular_velocities.resizeAndFill(n_particles, 0.0);
    torques.resizeAndFill(n_particles, 0.0);
    particle_start_index.resizeAndFill(n_particles, 0);
    num_vertices_in_particle.resizeAndFill(n_particles, 0);
}

void RigidBumpy::initGeometricVariables() {
    area.resizeAndFill(n_particles, 0.0);
}

void RigidBumpy::clearDynamicVariables() {
    Particle::clearDynamicVariables();
    vertex_positions.clear();
    vertex_velocities.clear();
    vertex_forces.clear();
    vertex_torques.clear();
    vertex_potential_energy.clear();
    angles.clear();
    angular_velocities.clear();
    torques.clear();
    vertex_particle_index.clear();
    particle_start_index.clear();
    num_vertices_in_particle.clear();
}

void RigidBumpy::syncVertexIndices() {
    hipError_t cuda_err = hipMemcpyToSymbol(HIP_SYMBOL(d_particle_start_index_ptr), &particle_start_index.d_ptr, sizeof(particle_start_index.d_ptr));
    if (cuda_err != hipSuccess) {
        std::cerr << "RigidBumpy::syncVertexIndices: Error copying d_particle_start_index_ptr to device: " << hipGetErrorString(cuda_err) << std::endl;
        exit(EXIT_FAILURE);  // TODO: make this a function and put it in a cuda module
    }
    cuda_err = hipMemcpyToSymbol(HIP_SYMBOL(d_num_vertices_in_particle_ptr), &num_vertices_in_particle.d_ptr, sizeof(num_vertices_in_particle.d_ptr));
    if (cuda_err != hipSuccess) {
        std::cerr << "RigidBumpy::syncVertexIndices: Error copying d_num_vertices_in_particle_ptr to device: " << hipGetErrorString(cuda_err) << std::endl;
        exit(EXIT_FAILURE);  // TODO: make this a function and put it in a cuda module
    }
    cuda_err = hipMemcpyToSymbol(HIP_SYMBOL(d_vertex_particle_index_ptr), &vertex_particle_index.d_ptr, sizeof(vertex_particle_index.d_ptr));
    if (cuda_err != hipSuccess) {
        std::cerr << "RigidBumpy::syncVertexIndices: Error copying d_vertex_particle_index_ptr to device: " << hipGetErrorString(cuda_err) << std::endl;
        exit(EXIT_FAILURE);  // TODO: make this a function and put it in a cuda module
    }
}

void RigidBumpy::setParticleStartIndex() {
    thrust::exclusive_scan(num_vertices_in_particle.d_vec.begin(), num_vertices_in_particle.d_vec.end(), particle_start_index.d_vec.begin());
}

void RigidBumpy::syncVertexRadius(double vertex_radius) {
    hipMemcpyToSymbol(HIP_SYMBOL(d_vertex_radius), &vertex_radius, sizeof(double));
}

double RigidBumpy::getVertexRadius() {
    double vertex_radius;
    hipMemcpyFromSymbol(&vertex_radius, HIP_SYMBOL(d_vertex_radius), sizeof(double));
    return vertex_radius;
}


// void Particle::setBiDispersity(double size_ratio, double count_ratio) {
//     if (size_ratio < 1.0) {
//         throw std::invalid_argument("Particle::setBiDispersity: size_ratio must be > 1.0");
//     }
//     if (count_ratio < 0.0 || count_ratio > 1.0) {
//         throw std::invalid_argument("Particle::setBiDispersity: count_ratio must be < 1.0 and > 0.0");
//     }
//     thrust::host_vector<double> host_radii(n_particles);
//     long n_large = static_cast<long>(n_particles * count_ratio);
//     double diam_large = size_ratio;
//     double diam_small = 1.0;
//     for (long i = 0; i < n_large; i++) {
//         host_radii[i] = diam_large / 2.0;
//     }
//     for (long i = n_large; i < n_particles; i++) {
//         host_radii[i] = diam_small / 2.0;
//     }
//     radii.setData(host_radii);
// }
long RigidBumpy::setVertexBiDispersity(long num_vertices_in_small_particle) {
    double max_particle_diam = getDiameter("max");
    double min_particle_diam = getDiameter("min");

    auto num_small_particles = thrust::count_if(
        radii.d_vec.begin(), 
        radii.d_vec.end(), 
        [=] __device__ (double rad) {
            return rad == min_particle_diam / 2.0;
        }
    );

    auto num_large_particles = n_particles - num_small_particles;
    long num_vertices_in_large_particle = static_cast<long>(num_vertices_in_small_particle * max_particle_diam / min_particle_diam);

    double vertex_angle_small = 2 * M_PI / num_vertices_in_small_particle;
    double vertex_radius = min_particle_diam / (1 + segment_length_per_vertex_diameter / std::sin(vertex_angle_small / 2)) / 2.0;
    syncVertexRadius(vertex_radius);
    setNumVertices(num_small_particles * num_vertices_in_small_particle + num_large_particles * num_vertices_in_large_particle);
    return num_vertices_in_large_particle;
}

void RigidBumpy::setDegreesOfFreedom() {
    this->n_dof = n_particles * (N_DIM + 1);  // two translation and one rotation
}

void RigidBumpy::initializeVerticesFromDiskPacking(SwapData2D<double>& disk_positions, SwapData1D<double>& disk_radii, long num_vertices_in_small_particle, long particle_dim_block, long vertex_dim_block) {
    // set the number of particles from the disk data
    setNumParticles(disk_positions.size[0]);
    initDynamicVariables();
    initGeometricVariables();

    // set the particle positions and radii from the disk packing
    positions.copyFrom(disk_positions);
    radii.copyFrom(disk_radii);


    // define the number of vertices using the bidispersity
    long num_vertices_in_large_particle = setVertexBiDispersity(num_vertices_in_small_particle);

    setDegreesOfFreedom();

    // set the kernel dimensions
    setKernelDimensions(particle_dim_block, vertex_dim_block);

    // initialize the vertex variables
    initVertexVariables();

    double min_particle_diam = getDiameter("min");
    double max_particle_diam = getDiameter("max");

    std::cout << "min_particle_diam: " << min_particle_diam << std::endl;
    std::cout << "max_particle_diam: " << max_particle_diam << std::endl;
    std::cout << "num_vertices_in_small_particle: " << num_vertices_in_small_particle << std::endl;
    std::cout << "num_vertices_in_large_particle: " << num_vertices_in_large_particle << std::endl;

    // set the number of vertices in each particle
    kernelGetNumVerticesInParticles<<<particle_dim_grid, particle_dim_block>>>(
        radii.d_ptr, min_particle_diam, num_vertices_in_small_particle, max_particle_diam, num_vertices_in_large_particle, num_vertices_in_particle.d_ptr);

    // set the particle start index
    setParticleStartIndex();

    // initialize the vertices on the particles
    kernelInitializeVerticesOnParticles<<<particle_dim_grid, particle_dim_block>>>(
        positions.x.d_ptr, positions.y.d_ptr, radii.d_ptr, angles.d_ptr, vertex_particle_index.d_ptr, particle_start_index.d_ptr, num_vertices_in_particle.d_ptr, vertex_masses.d_ptr, vertex_positions.x.d_ptr, vertex_positions.y.d_ptr);
    
    // sync the vertex indices
    syncVertexIndices();
}




ArrayData RigidBumpy::getArrayData(const std::string& array_name) {
    try {
        return Particle::getArrayData(array_name);
    } catch (std::invalid_argument& e) {
        // try the rigid bumpy specific ones
        ArrayData result;
        result.name = array_name;
        if (array_name == "vertex_positions") {
            result.type = DataType::Double;
            result.size = vertex_positions.size;
            result.data = std::make_pair(vertex_positions.getDataX(), vertex_positions.getDataY());
            result.index_array_name = "static_vertex_index";
        } else if (array_name == "vertex_velocities") {
            result.type = DataType::Double;
            result.size = vertex_velocities.size;
            result.data = std::make_pair(vertex_velocities.getDataX(), vertex_velocities.getDataY());
            result.index_array_name = "static_vertex_index";
        } else if (array_name == "vertex_forces") {
            result.type = DataType::Double;
            result.size = vertex_forces.size;
            result.data = std::make_pair(vertex_forces.getDataX(), vertex_forces.getDataY());
            result.index_array_name = "static_vertex_index";
        } else if (array_name == "vertex_masses") {
            result.type = DataType::Double;
            result.size = vertex_masses.size;
            result.data = vertex_masses.getData();
            result.index_array_name = "static_vertex_index";
        } else if (array_name == "angles") {
            result.type = DataType::Double;
            result.size = angles.size;
            result.data = angles.getData();
            result.index_array_name = "static_particle_index";
        } else if (array_name == "angular_velocities") {
            result.type = DataType::Double;
            result.size = angular_velocities.size;
            result.data = angular_velocities.getData();
            result.index_array_name = "static_particle_index";
        } else if (array_name == "torques") {
            result.type = DataType::Double;
            result.size = torques.size;
            result.data = torques.getData();
            result.index_array_name = "static_particle_index";
        } else if (array_name == "area") {
            result.type = DataType::Double;
            result.size = area.size;
            result.data = area.getData();
            result.index_array_name = "static_particle_index";
        } else if (array_name == "vertex_particle_index") {
            result.type = DataType::Long;
            result.size = vertex_particle_index.size;
            result.data = vertex_particle_index.getData();
            result.index_array_name = "static_vertex_index";
        } else if (array_name == "particle_start_index") {
            result.type = DataType::Long;
            result.size = particle_start_index.size;
            result.data = particle_start_index.getData();
            result.index_array_name = "static_particle_index";
        } else if (array_name == "num_vertices_in_particle") {
            result.type = DataType::Long;
            result.size = num_vertices_in_particle.size;
            result.data = num_vertices_in_particle.getData();
            result.index_array_name = "static_particle_index";
        } else if (array_name == "vertex_neighbor_list") {
            result.type = DataType::Long;
            result.size = vertex_neighbor_list.size;
            result.data = vertex_neighbor_list.getData();
            result.index_array_name = "";// TODO:
        } else if (array_name == "num_vertex_neighbors") {
            result.type = DataType::Long;
            result.size = num_vertex_neighbors.size;
            result.data = num_vertex_neighbors.getData();
            result.index_array_name = "static_vertex_index";
        } else if (array_name == "vertex_index") {
            result.type = DataType::Long;
            result.size = vertex_index.size;
            result.data = vertex_index.getData();
            result.index_array_name = "static_vertex_index";
        } else if (array_name == "static_vertex_index") {
            result.type = DataType::Long;
            result.size = static_vertex_index.size;
            result.data = static_vertex_index.getData();
            result.index_array_name = "";

        } else {
            throw std::invalid_argument("RigidBumpy::getArrayData: array_name " + array_name + " not found");
        }
    }
}





// need to make a scale function for the particles which can then go into the base particle class and be overridden by the rigid bumpy class so we dont have to replicate the scaleToPackingFraction function

void RigidBumpy::calculateParticleArea() {
    // kernelCalculateParticlePolygonArea<<<particle_dim_grid, particle_dim_block>>>(
    //     vertex_positions.x.d_ptr, vertex_positions.y.d_ptr, area.d_ptr);
    kernelCalculateBumpyParticleAreaFull<<<particle_dim_grid, particle_dim_block>>>(
        vertex_positions.x.d_ptr, vertex_positions.y.d_ptr, radii.d_ptr, area.d_ptr);
}

double RigidBumpy::getParticleArea() const {
    double a = thrust::reduce(area.d_vec.begin(), area.d_vec.end(), 0.0, thrust::plus<double>());
    double box_area = getBoxArea();
    return a;
}


// calculate the particle positions first if not already done not needed for rigid bumpy
// compare polygon areas to particle areas
// calculate the contribution to the area from the vertices
void RigidBumpy::scalePositions(double scale_factor) {
    kernelScalePositions<<<particle_dim_grid, particle_dim_block>>>(
        positions.x.d_ptr, positions.y.d_ptr, vertex_positions.x.d_ptr, vertex_positions.y.d_ptr, scale_factor
    );
}

void RigidBumpy::syncVertexNeighborList() {
    hipError_t cuda_err = hipMemcpyToSymbol(HIP_SYMBOL(d_max_vertex_neighbors_allocated), &max_vertex_neighbors_allocated, sizeof(max_vertex_neighbors_allocated));
    if (cuda_err != hipSuccess) {
        std::cerr << "RigidBumpy::syncVertexNeighborList: Error copying max_vertex_neighbors_allocated to device: " << hipGetErrorString(cuda_err) << std::endl;
        exit(EXIT_FAILURE);  // TODO: make this a function and put it in a cuda module
    }
    cuda_err = hipMemcpyToSymbol(HIP_SYMBOL(d_vertex_neighbor_list_ptr), &vertex_neighbor_list.d_ptr, sizeof(vertex_neighbor_list.d_ptr));
    if (cuda_err != hipSuccess) {
        std::cerr << "RigidBumpy::syncVertexNeighborList: Error copying d_vertex_neighbor_list_ptr to device: " << hipGetErrorString(cuda_err) << std::endl;
        exit(EXIT_FAILURE);  // TODO: make this a function and put it in a cuda module
    }
    cuda_err = hipMemcpyToSymbol(HIP_SYMBOL(d_num_vertex_neighbors_ptr), &num_vertex_neighbors.d_ptr, sizeof(num_vertex_neighbors.d_ptr));
    if (cuda_err != hipSuccess) {
        std::cerr << "RigidBumpy::syncVertexNeighborList: Error copying d_num_vertex_neighbors_ptr to device: " << hipGetErrorString(cuda_err) << std::endl;
        exit(EXIT_FAILURE);  // TODO: make this a function and put it in a cuda module
    }
}

void RigidBumpy::setMass(double mass) {
    Particle::setMass(mass);
    vertex_masses.scale(mass);
    // check if sum of vertex masses is equal to particle mass for each particle
    double total_vertex_mass = thrust::reduce(vertex_masses.d_vec.begin(), vertex_masses.d_vec.end(), 0.0, thrust::plus<double>());
    if (std::abs(total_vertex_mass / n_particles - mass) > 1e-6) {
        std::cout << "WARNING: RigidBumpy::setMass: Total vertex mass does not match particle mass" << std::endl;
    }
}

double RigidBumpy::getOverlapFraction() const {
    std::cout << "FIXME: Implement getOverlapFraction" << std::endl;
    return 0.0;
}

void RigidBumpy::calculateForces() {
    // version 1: 2 kernels, 1 vertex level and 1 particle level
    kernelCalcRigidBumpyForces1<<<vertex_dim_grid, vertex_dim_block>>>(
        positions.x.d_ptr, positions.y.d_ptr, vertex_positions.x.d_ptr, vertex_positions.y.d_ptr, vertex_forces.x.d_ptr, vertex_forces.y.d_ptr, vertex_torques.d_ptr, vertex_potential_energy.d_ptr
    );
    kernelCalcRigidBumpyParticleForces1<<<particle_dim_grid, particle_dim_block>>>(
        vertex_forces.x.d_ptr, vertex_forces.y.d_ptr, vertex_torques.d_ptr, vertex_potential_energy.d_ptr, forces.x.d_ptr, forces.y.d_ptr, torques.d_ptr, potential_energy.d_ptr
    );

    // version 2: 1 particle level kernel
    // kernelCalcRigidBumpyForces2<<<particle_dim_grid, particle_dim_block>>>(
    //     positions.x.d_ptr, positions.y.d_ptr, vertex_positions.x.d_ptr, vertex_positions.y.d_ptr, forces.x.d_ptr, forces.y.d_ptr, torques.d_ptr, potential_energy.d_ptr, vertex_forces.x.d_ptr, vertex_forces.y.d_ptr, vertex_torques.d_ptr, vertex_potential_energy.d_ptr
    // );
}

void RigidBumpy::updatePositions() {
}

void RigidBumpy::updateVelocities() {
}

void RigidBumpy::calculateKineticEnergy() {
}

void RigidBumpy::calculateParticlePositions() {
    kernelCalculateParticlePositions<<<particle_dim_grid, particle_dim_block>>>(
        vertex_positions.x.d_ptr, vertex_positions.y.d_ptr, positions.x.d_ptr, positions.y.d_ptr
    );
}

void RigidBumpy::updateVertexVerletList() {
    std::cout << "Updating vertex verlet list" << std::endl;
    vertex_neighbor_list.fill(-1L);
    kernelUpdateVertexNeighborList<<<vertex_dim_grid, vertex_dim_block>>>(
        vertex_positions.x.d_ptr, vertex_positions.y.d_ptr, positions.x.d_ptr, positions.y.d_ptr, vertex_neighbor_cutoff, vertex_particle_neighbor_cutoff
    );
    long max_vertex_neighbors = thrust::reduce(num_vertex_neighbors.d_vec.begin(), num_vertex_neighbors.d_vec.end(), -1L, thrust::maximum<long>());
    std::cout << "max_vertex_neighbors: " << max_vertex_neighbors << std::endl;
    if (max_vertex_neighbors > max_vertex_neighbors_allocated) {
        max_vertex_neighbors_allocated = std::pow(2, std::ceil(std::log2(max_vertex_neighbors)));
        std::cout << "RigidBumpy::updateVertexVerletList: Resizing vertex neighbor list to " << max_vertex_neighbors_allocated << std::endl;
        vertex_neighbor_list.resize(n_vertices * max_vertex_neighbors_allocated);
        vertex_neighbor_list.fill(-1L);
        syncVertexNeighborList();
        kernelUpdateVertexNeighborList<<<vertex_dim_grid, vertex_dim_block>>>(
            vertex_positions.x.d_ptr, vertex_positions.y.d_ptr, positions.x.d_ptr, positions.y.d_ptr, vertex_neighbor_cutoff, vertex_particle_neighbor_cutoff
        );
    }
}

void RigidBumpy::initVerletListVariables() {
    Particle::initVerletListVariables();
    vertex_neighbor_list.resizeAndFill(n_vertices * max_vertex_neighbors_allocated, -1L);
    num_vertex_neighbors.resizeAndFill(n_vertices, 0L);
}

void RigidBumpy::initVerletList() {
    initVerletListVariables();
    syncNeighborList();
    syncVertexNeighborList();
    updateVerletList();
    updateVertexVerletList();
}

