#include "hip/hip_runtime.h"
#include "../../include/constants.h"
#include "../../include/cuda_constants.cuh"
#include "../../include/functors.h"
#include "../../include/particle/particle.h"
#include "../../include/kernels/dynamics.cuh"
#include "../../include/kernels/contacts.cuh"
#include <iostream>
#include <string>
#include <vector>
#include <cmath>
#include <random>
#include <time.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/fill.h>
#include <thrust/copy.h>
#include <thrust/transform.h>
#include <thrust/transform_reduce.h>
#include <thrust/inner_product.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/permutation_iterator.h>
#include <thrust/functional.h>

// Constructor
Particle::Particle() {
}

// Destructor (virtual to allow proper cleanup in derived classes)
Particle::~Particle() {
}

// ----------------------------------------------------------------------
// ----------------------- Template Methods -----------------------------
// ----------------------------------------------------------------------

std::unordered_map<std::string, std::any> Particle::getArrayMap() {
    std::unordered_map<std::string, std::any> array_map;
    array_map["d_positions"]        = &d_positions;
    array_map["d_last_positions"]   = &d_last_positions;
    array_map["d_velocities"]       = &d_velocities;
    array_map["d_forces"]           = &d_forces;
    array_map["d_radii"]            = &d_radii;
    array_map["d_masses"]           = &d_masses;
    array_map["d_potential_energy"] = &d_potential_energy;
    array_map["d_kinetic_energy"]   = &d_kinetic_energy;
    array_map["d_neighbor_list"]    = &d_neighbor_list;
    array_map["d_num_neighbors"]  = &d_num_neighbors;
    return array_map;
}

// ----------------------------------------------------------------------
// -------------------- Universally Defined Methods ---------------------
// ----------------------------------------------------------------------

void Particle::setSeed(long seed) {
    if (seed == -1) {
        seed = time(0);
    }
    this->seed = seed;
    srand(seed);
}

void Particle::setNumParticles(long n_particles) {
    this->n_particles = n_particles;
    this->n_dof = n_particles * N_DIM;
}

void Particle::setNumVertices(long n_vertices) {
    this->n_vertices = n_vertices;
}

void Particle::setKernelDimensions(long dim_block) {
    int maxThreadsPerBlock;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    maxThreadsPerBlock = deviceProp.maxThreadsPerBlock;
    std::cout << "CUDA Info: Max threads per block: " << maxThreadsPerBlock << std::endl;

    // Ensure dim_block doesn't exceed the maxThreadsPerBlock
    if (dim_block > maxThreadsPerBlock) {
        std::cout << "WARNING: dim_block exceeds maxThreadsPerBlock, adjusting to maxThreadsPerBlock" << std::endl;
        dim_block = maxThreadsPerBlock;
    }

    // If there are few particles, set dim_block to the number of particles
    if (n_particles < dim_block) {
        dim_block = n_particles;
    }

    if (n_particles <= 0) {
        std::cout << "WARNING: Particle::setKernelDimensions: n_particles is 0.  Ignore if only using vertex-level kernels, otherwise set n_particles." << std::endl;
        n_particles = 1;
    }
    if (n_vertices <= 0) {
        std::cout << "WARNING: Particle::setKernelDimensions: n_vertices is 0.  Ignore if only using particle-level kernels, otherwise set n_vertices." << std::endl;
        n_vertices = 1;
    }

    // Set block and grid dimensions for particles
    this->dim_block = dim_block;
    this->dim_grid = (n_particles + dim_block - 1) / dim_block;

    // Set block and grid dimensions for vertices (optional)
    this->dim_vertex_grid = (n_vertices + dim_block - 1) / dim_block;
}

void Particle::initDynamicVariables() {
    // Resize the device vectors
    d_positions.resize(n_particles * N_DIM);
    d_last_positions.resize(n_particles * N_DIM);
    d_displacements.resize(n_particles * N_DIM);
    d_velocities.resize(n_particles * N_DIM);
    d_forces.resize(n_particles * N_DIM);
    d_radii.resize(n_particles);
    d_masses.resize(n_particles);
    d_potential_energy.resize(n_particles);
    d_kinetic_energy.resize(n_particles);
    d_neighbor_list.resize(n_particles);
    d_num_neighbors.resize(n_particles);
    max_neighbors = 0;
    max_neighbors_allocated = 0;
    thrust::fill(d_neighbor_list.begin(), d_neighbor_list.end(), -1L);
    thrust::fill(d_num_neighbors.begin(), d_num_neighbors.end(), max_neighbors);


    // Cast the raw pointers
    d_positions_ptr = thrust::raw_pointer_cast(&d_positions[0]);
    d_last_positions_ptr = thrust::raw_pointer_cast(&d_last_positions[0]);
    d_displacements_ptr = thrust::raw_pointer_cast(&d_displacements[0]);
    d_velocities_ptr = thrust::raw_pointer_cast(&d_velocities[0]);
    d_forces_ptr = thrust::raw_pointer_cast(&d_forces[0]);
    d_radii_ptr = thrust::raw_pointer_cast(&d_radii[0]);
    d_masses_ptr = thrust::raw_pointer_cast(&d_masses[0]);
    d_potential_energy_ptr = thrust::raw_pointer_cast(&d_potential_energy[0]);
    d_kinetic_energy_ptr = thrust::raw_pointer_cast(&d_kinetic_energy[0]);
}

void Particle::clearDynamicVariables() {
    // Clear the device vectors
    d_positions.clear();
    d_last_positions.clear();
    d_displacements.clear();
    d_velocities.clear();
    d_forces.clear();
    d_radii.clear();
    d_masses.clear();
    d_potential_energy.clear();
    d_kinetic_energy.clear();
    d_neighbor_list.clear();
    d_num_neighbors.clear();

    // Clear the pointers
    d_positions_ptr = nullptr;
    d_last_positions_ptr = nullptr;
    d_displacements_ptr = nullptr;
    d_velocities_ptr = nullptr;
    d_forces_ptr = nullptr;
    d_radii_ptr = nullptr;
    d_masses_ptr = nullptr;
    d_potential_energy_ptr = nullptr;
    d_kinetic_energy_ptr = nullptr;
}

void Particle::setBoxSize(const thrust::host_vector<double>& box_size) {
    if (box_size.size() != N_DIM) {
        throw std::invalid_argument("Particle::setBoxSize: Error box_size (" + std::to_string(box_size.size()) + ")" + " != " + std::to_string(N_DIM) + " elements");
    }
    hipError_t cuda_err = hipMemcpyToSymbol(HIP_SYMBOL(d_box_size), box_size.data(), sizeof(double) * N_DIM);
    if (cuda_err != hipSuccess) {
        std::cerr << "Particle::setBoxSize: Error copying box size to device: " << hipGetErrorString(cuda_err) << std::endl;
        exit(EXIT_FAILURE);
    }
}

thrust::host_vector<double> Particle::getBoxSize() {
    thrust::host_vector<double> box_size(N_DIM);
    hipError_t cuda_err = hipMemcpyFromSymbol(&box_size[0], HIP_SYMBOL(d_box_size), sizeof(double) * N_DIM);
    for (int i = 0; i < N_DIM; i++) {
        std::cout << "Particle::getBoxSize: d_box_size[" << i << "]: " << box_size[i] << std::endl;
    }
    if (cuda_err != hipSuccess) {
        std::cerr << "Particle::getBoxSize: Error copying box size to host: " << hipGetErrorString(cuda_err) << std::endl;
        exit(EXIT_FAILURE);
    }
    return box_size;
}

void Particle::syncNeighborList() {
    hipError_t cuda_err = hipMemcpyToSymbol(HIP_SYMBOL(d_max_neighbors), &this->max_neighbors, sizeof(this->max_neighbors));
    if (cuda_err != hipSuccess) {
        std::cerr << "Particle::syncNeighborList: Error copying max_neighbors to device: " << hipGetErrorString(cuda_err) << std::endl;
        exit(EXIT_FAILURE);
    }
    cuda_err = hipMemcpyToSymbol(HIP_SYMBOL(d_max_neighbors_allocated), &this->max_neighbors_allocated, sizeof(this->max_neighbors_allocated));
    if (cuda_err != hipSuccess) {
        std::cerr << "Particle::syncNeighborList: Error copying max_neighbors_allocated to device: " << hipGetErrorString(cuda_err) << std::endl;
        exit(EXIT_FAILURE);
    }
    long* neighbor_list_ptr = thrust::raw_pointer_cast(&d_neighbor_list[0]);
    cuda_err = hipMemcpyToSymbol(HIP_SYMBOL(d_neighbor_list_ptr), &neighbor_list_ptr, sizeof(neighbor_list_ptr));
    if (cuda_err != hipSuccess) {
        std::cerr << "Particle::syncNeighborList: Error copying d_neighbor_list_ptr to device: " << hipGetErrorString(cuda_err) << std::endl;
        exit(EXIT_FAILURE);
    }
    long* num_neighbors_ptr = thrust::raw_pointer_cast(&d_num_neighbors[0]);
    cuda_err = hipMemcpyToSymbol(HIP_SYMBOL(d_num_neighbors_ptr), &num_neighbors_ptr, sizeof(num_neighbors_ptr));
    if (cuda_err != hipSuccess) {
        std::cerr << "Particle::syncNeighborList: Error copying d_num_neighbors_ptr to device: " << hipGetErrorString(cuda_err) << std::endl;
        exit(EXIT_FAILURE);
    }
}

void Particle::setEnergyScale(double e, std::string which) {
    if (which == "c") {
        e_c = e;
    } else if (which == "a") {
        e_a = e;
    } else if (which == "b") {
        e_b = e;
    } else if (which == "l") {
        e_l = e;
    } else {
        throw std::invalid_argument("Particle::setEnergyScale: which must be 'c', 'a', 'b', or 'l', not " + which);
    }
}

void Particle::setExponent(double n, std::string which) {
    if (which == "c") {
        n_c = n;
    } else if (which == "a") {
        n_a = n;
    } else if (which == "b") {
        n_b = n;
    } else if (which == "l") {
        n_l = n;
    } else {
        throw std::invalid_argument("Particle::setExponent: which must be 'c', 'a', 'b', or 'l', not " + which);
    }
}

void Particle::setCudaConstants() {
    hipError_t cuda_err;
    cuda_err = hipMemcpyToSymbol(HIP_SYMBOL(d_n_particles), &n_particles, sizeof(long));
    if (cuda_err != hipSuccess) {
        std::cerr << "Particle::setCudaConstants: Error copying n_particles to device: " << hipGetErrorString(cuda_err) << std::endl;
        exit(EXIT_FAILURE);
    }
    cuda_err = hipMemcpyToSymbol(HIP_SYMBOL(d_n_vertices), &n_vertices, sizeof(long));
    if (cuda_err != hipSuccess) {
        std::cerr << "Particle::setCudaConstants: Error copying n_vertices to device: " << hipGetErrorString(cuda_err) << std::endl;
        exit(EXIT_FAILURE);
    }

    cuda_err = hipMemcpyToSymbol(HIP_SYMBOL(d_dim_block), &dim_block, sizeof(long));
    if (cuda_err != hipSuccess) {
        std::cerr << "Particle::setCudaConstants: Error copying dim_block to device: " << hipGetErrorString(cuda_err) << std::endl;
        exit(EXIT_FAILURE);
    }
    cuda_err = hipMemcpyToSymbol(HIP_SYMBOL(d_dim_grid), &dim_grid, sizeof(long));
    if (cuda_err != hipSuccess) {
        std::cerr << "Particle::setCudaConstants: Error copying dim_grid to device: " << hipGetErrorString(cuda_err) << std::endl;
        exit(EXIT_FAILURE);
    }
    cuda_err = hipMemcpyToSymbol(HIP_SYMBOL(d_dim_vertex_grid), &dim_vertex_grid, sizeof(long));
    if (cuda_err != hipSuccess) {
        std::cerr << "Particle::setCudaConstants: Error copying dim_vertex_grid to device: " << hipGetErrorString(cuda_err) << std::endl;
        exit(EXIT_FAILURE);
    }
    cuda_err = hipMemcpyToSymbol(HIP_SYMBOL(d_max_neighbors), &max_neighbors, sizeof(long));
    if (cuda_err != hipSuccess) {
        std::cerr << "Particle::setCudaConstants: Error copying max_neighbors to device: " << hipGetErrorString(cuda_err) << std::endl;
        exit(EXIT_FAILURE);
    }
    cuda_err = hipMemcpyToSymbol(HIP_SYMBOL(d_max_neighbors_allocated), &max_neighbors_allocated, sizeof(long));
    if (cuda_err != hipSuccess) {
        std::cerr << "Particle::setCudaConstants: Error copying max_neighbors_allocated to device: " << hipGetErrorString(cuda_err) << std::endl;
        exit(EXIT_FAILURE);
    }

    cuda_err = hipMemcpyToSymbol(HIP_SYMBOL(d_e_c), &e_c, sizeof(double));
    if (cuda_err != hipSuccess) {
        std::cerr << "Particle::setCudaConstants: Error copying e_c to device: " << hipGetErrorString(cuda_err) << std::endl;
        exit(EXIT_FAILURE);
    }
    cuda_err = hipMemcpyToSymbol(HIP_SYMBOL(d_e_a), &e_a, sizeof(double));
    if (cuda_err != hipSuccess) {
        std::cerr << "Particle::setCudaConstants: Error copying e_a to device: " << hipGetErrorString(cuda_err) << std::endl;
        exit(EXIT_FAILURE);
    }
    cuda_err = hipMemcpyToSymbol(HIP_SYMBOL(d_e_b), &e_b, sizeof(double));
    if (cuda_err != hipSuccess) {
        std::cerr << "Particle::setCudaConstants: Error copying e_b to device: " << hipGetErrorString(cuda_err) << std::endl;
        exit(EXIT_FAILURE);
    }
    cuda_err = hipMemcpyToSymbol(HIP_SYMBOL(d_e_l), &e_l, sizeof(double));
    if (cuda_err != hipSuccess) {
        std::cerr << "Particle::setCudaConstants: Error copying e_l to device: " << hipGetErrorString(cuda_err) << std::endl;
        exit(EXIT_FAILURE);
    }
    cuda_err = hipMemcpyToSymbol(HIP_SYMBOL(d_n_c), &n_c, sizeof(double));
    if (cuda_err != hipSuccess) {
        std::cerr << "Particle::setCudaConstants: Error copying n_c to device: " << hipGetErrorString(cuda_err) << std::endl;
        exit(EXIT_FAILURE);
    }
    cuda_err = hipMemcpyToSymbol(HIP_SYMBOL(d_n_a), &n_a, sizeof(double));
    if (cuda_err != hipSuccess) {
        std::cerr << "Particle::setCudaConstants: Error copying n_a to device: " << hipGetErrorString(cuda_err) << std::endl;
        exit(EXIT_FAILURE);
    }
    cuda_err = hipMemcpyToSymbol(HIP_SYMBOL(d_n_b), &n_b, sizeof(double));
    if (cuda_err != hipSuccess) {
        std::cerr << "Particle::setCudaConstants: Error copying n_b to device: " << hipGetErrorString(cuda_err) << std::endl;
        exit(EXIT_FAILURE);
    }
    cuda_err = hipMemcpyToSymbol(HIP_SYMBOL(d_n_l), &n_l, sizeof(double));
    if (cuda_err != hipSuccess) {
        std::cerr << "Particle::setCudaConstants: Error copying n_l to device: " << hipGetErrorString(cuda_err) << std::endl;
        exit(EXIT_FAILURE);
    }
}

void Particle::getCudaConstants() {
    hipError_t cuda_err;
    long temp;
    cuda_err = hipMemcpyFromSymbol(&temp, HIP_SYMBOL(d_n_particles), sizeof(long));
    if (cuda_err != hipSuccess) {
        std::cerr << "Particle::getCudaConstants: Error copying n_particles from device: " << hipGetErrorString(cuda_err) << std::endl;
        exit(EXIT_FAILURE);
    }
    std::cout << "n_particles: " << temp << std::endl;
    cuda_err = hipMemcpyFromSymbol(&temp, HIP_SYMBOL(d_n_vertices), sizeof(long));
    if (cuda_err != hipSuccess) {
        std::cerr << "Particle::getCudaConstants: Error copying n_vertices from device: " << hipGetErrorString(cuda_err) << std::endl;
        exit(EXIT_FAILURE);
    }
    std::cout << "n_vertices: " << temp << std::endl;


    cuda_err = hipMemcpyFromSymbol(&dim_block, HIP_SYMBOL(d_dim_block), sizeof(long));
    if (cuda_err != hipSuccess) {
        std::cerr << "Particle::getCudaConstants: Error copying dim_block from device: " << hipGetErrorString(cuda_err) << std::endl;
        exit(EXIT_FAILURE);
    }
    std::cout << "dim_block: " << dim_block << std::endl;
    cuda_err = hipMemcpyFromSymbol(&dim_grid, HIP_SYMBOL(d_dim_grid), sizeof(long));
    if (cuda_err != hipSuccess) {
        std::cerr << "Particle::getCudaConstants: Error copying dim_grid from device: " << hipGetErrorString(cuda_err) << std::endl;
        exit(EXIT_FAILURE);
    }
    std::cout << "dim_grid: " << dim_grid << std::endl;
    cuda_err = hipMemcpyFromSymbol(&dim_vertex_grid, HIP_SYMBOL(d_dim_vertex_grid), sizeof(long));
    if (cuda_err != hipSuccess) {
        std::cerr << "Particle::getCudaConstants: Error copying dim_vertex_grid from device: " << hipGetErrorString(cuda_err) << std::endl;
        exit(EXIT_FAILURE);
    }
    std::cout << "dim_vertex_grid: " << dim_vertex_grid << std::endl;
    cuda_err = hipMemcpyFromSymbol(&max_neighbors, HIP_SYMBOL(d_max_neighbors), sizeof(long));
    if (cuda_err != hipSuccess) {
        std::cerr << "Particle::getCudaConstants: Error copying max_neighbors from device: " << hipGetErrorString(cuda_err) << std::endl;
        exit(EXIT_FAILURE);
    }
    std::cout << "max_neighbors: " << max_neighbors << std::endl;
    cuda_err = hipMemcpyFromSymbol(&max_neighbors_allocated, HIP_SYMBOL(d_max_neighbors_allocated), sizeof(long));
    if (cuda_err != hipSuccess) {
        std::cerr << "Particle::getCudaConstants: Error copying max_neighbors_allocated from device: " << hipGetErrorString(cuda_err) << std::endl;
        exit(EXIT_FAILURE);
    }
    std::cout << "max_neighbors_allocated: " << max_neighbors_allocated << std::endl;
    cuda_err = hipMemcpyFromSymbol(&e_c, HIP_SYMBOL(d_e_c), sizeof(double));
    if (cuda_err != hipSuccess) {
        std::cerr << "Particle::getCudaConstants: Error copying e_c from device: " << hipGetErrorString(cuda_err) << std::endl;
        exit(EXIT_FAILURE);
    }
    std::cout << "e_c: " << e_c << std::endl;
    cuda_err = hipMemcpyFromSymbol(&e_a, HIP_SYMBOL(d_e_a), sizeof(double));
    if (cuda_err != hipSuccess) {
        std::cerr << "Particle::getCudaConstants: Error copying e_a from device: " << hipGetErrorString(cuda_err) << std::endl;
        exit(EXIT_FAILURE);
    }
    std::cout << "e_a: " << e_a << std::endl;
    cuda_err = hipMemcpyFromSymbol(&e_b, HIP_SYMBOL(d_e_b), sizeof(double));
    if (cuda_err != hipSuccess) {
        std::cerr << "Particle::getCudaConstants: Error copying e_b from device: " << hipGetErrorString(cuda_err) << std::endl;
        exit(EXIT_FAILURE);
    }
    std::cout << "e_b: " << e_b << std::endl;
    cuda_err = hipMemcpyFromSymbol(&e_l, HIP_SYMBOL(d_e_l), sizeof(double));
    if (cuda_err != hipSuccess) {
        std::cerr << "Particle::getCudaConstants: Error copying e_l from device: " << hipGetErrorString(cuda_err) << std::endl;
        exit(EXIT_FAILURE);
    }
    std::cout << "e_l: " << e_l << std::endl;
    cuda_err = hipMemcpyFromSymbol(&n_c, HIP_SYMBOL(d_n_c), sizeof(double));
    if (cuda_err != hipSuccess) {
        std::cerr << "Particle::getCudaConstants: Error copying n_c from device: " << hipGetErrorString(cuda_err) << std::endl;
        exit(EXIT_FAILURE);
    }
    std::cout << "n_c: " << n_c << std::endl;
    cuda_err = hipMemcpyFromSymbol(&n_a, HIP_SYMBOL(d_n_a), sizeof(double));
    if (cuda_err != hipSuccess) {
        std::cerr << "Particle::getCudaConstants: Error copying n_a from device: " << hipGetErrorString(cuda_err) << std::endl;
        exit(EXIT_FAILURE);
    }
    std::cout << "n_a: " << n_a << std::endl;
    cuda_err = hipMemcpyFromSymbol(&n_b, HIP_SYMBOL(d_n_b), sizeof(double));
    if (cuda_err != hipSuccess) {
        std::cerr << "Particle::getCudaConstants: Error copying n_b from device: " << hipGetErrorString(cuda_err) << std::endl;
        exit(EXIT_FAILURE);
    }
    std::cout << "n_b: " << n_b << std::endl;
    cuda_err = hipMemcpyFromSymbol(&n_l, HIP_SYMBOL(d_n_l), sizeof(double));
    if (cuda_err != hipSuccess) {
        std::cerr << "Particle::getCudaConstants: Error copying n_l from device: " << hipGetErrorString(cuda_err) << std::endl;
        exit(EXIT_FAILURE);
    }
    std::cout << "n_l: " << n_l << std::endl;
}

void Particle::initializeBox(double area) {
    double side_length = std::pow(area, 1.0 / N_DIM);
    thrust::host_vector<double> box_size(N_DIM, side_length);
    setBoxSize(box_size);
}

void Particle::setRandomUniform(thrust::device_vector<double>& values, double min, double max) {
    thrust::counting_iterator<long> index_sequence_begin(seed);
    thrust::transform(index_sequence_begin, index_sequence_begin + values.size(), values.begin(), RandomUniform(min, max, seed));
}

void Particle::setRandomNormal(thrust::device_vector<double>& values, double mean, double stddev) {
    thrust::counting_iterator<long> index_sequence_begin(seed);
    thrust::transform(index_sequence_begin, index_sequence_begin + values.size(), values.begin(), RandomNormal(mean, stddev, seed));
}

void Particle::setRandomPositions() {
    thrust::host_vector<double> box_size = getBoxSize();
    setRandomUniform(d_positions, 0.0, box_size[0]);
}

double Particle::getDiameter(std::string which) {
    if (which == "min") {
        return 2.0 * *thrust::min_element(d_radii.begin(), d_radii.end());
    } else if (which == "max") {
        return 2.0 * *thrust::max_element(d_radii.begin(), d_radii.end());
    } else if (which == "mean") {
        return 2.0 * thrust::reduce(d_radii.begin(), d_radii.end()) / d_radii.size();
    } else {
        throw std::invalid_argument("Particle::getDiameter: which must be 'min', 'max', or 'mean', not " + which);
    }
}

void Particle::setBiDispersity(double size_ratio, double count_ratio) {
    if (size_ratio < 1.0) {
        throw std::invalid_argument("Particle::setBiDispersity: size_ratio must be > 1.0");
    }
    if (count_ratio < 0.0 || count_ratio > 1.0) {
        throw std::invalid_argument("Particle::setBiDispersity: count_ratio must be < 1.0 and > 0.0");
    }
    thrust::host_vector<double> radii(n_particles);
    long n_large = static_cast<long>(n_particles * count_ratio);
    double r_large = size_ratio;
    double r_small = 1.0;
    for (long i = 0; i < n_large; i++) {
        radii[i] = r_large / 2.0;
    }
    for (long i = n_large; i < n_particles; i++) {
        radii[i] = r_small / 2.0;
    }
    setArray("d_radii", radii);
}

double Particle::getBoxArea() {
    thrust::host_vector<double> box_size = getBoxSize();
    return thrust::reduce(box_size.begin(), box_size.end(), 1.0, thrust::multiplies<double>());
}

double Particle::getPackingFraction() {
    double box_area = getBoxArea();
    double area = getArea();
    return area / box_area;
}

double Particle::getDensity() {
    return getPackingFraction() - getOverlapFraction();
}

void Particle::scaleToPackingFraction(double packing_fraction) {
    double new_side_length = std::pow(getArea() / packing_fraction, 1.0 / N_DIM);
    double side_length = std::pow(getBoxArea(), 1.0 / N_DIM);
    scalePositions(new_side_length / side_length);
    setBoxSize(thrust::host_vector<double>(N_DIM, new_side_length));
}

double Particle::totalKineticEnergy() const {
    thrust::host_vector<double> h_kinetic_energy = d_kinetic_energy;
    return thrust::reduce(h_kinetic_energy.begin(), h_kinetic_energy.end(), 0.0, thrust::plus<double>());
}

double Particle::totalPotentialEnergy() const {
    return thrust::reduce(d_potential_energy.begin(), d_potential_energy.end(), 0.0, thrust::plus<double>());
}

double Particle::totalEnergy() const {
    return totalKineticEnergy() + totalPotentialEnergy();
}

void Particle::scalePositions(double scale_factor) {
    thrust::transform(d_positions.begin(), d_positions.end(), thrust::make_constant_iterator(scale_factor), d_positions.begin(), thrust::multiplies<double>());
}

void Particle::updatePositions(double dt) {
    kernelUpdatePositions<<<dim_grid, dim_block>>>(d_positions_ptr, d_last_positions_ptr, d_displacements_ptr, d_velocities_ptr, dt);
}

void Particle::updateVelocities(double dt) {
    kernelUpdateVelocities<<<dim_grid, dim_block>>>(d_velocities_ptr, d_forces_ptr, d_masses_ptr, dt);
}

double Particle::getMaxDisplacement() {
    return thrust::reduce(d_displacements.begin(), d_displacements.end(), 0.0, thrust::maximum<double>());
}

void Particle::updateNeighborList() {
    std::cout << "dim_grid: " << dim_grid << std::endl;
    std::cout << "dim_block: " << dim_block << std::endl;
    std::cout << "n_particles: " << n_particles << std::endl;

    thrust::fill(d_num_neighbors.begin(), d_num_neighbors.end(), 0);
    thrust::fill(d_neighbor_list.begin(), d_neighbor_list.end(), -1L);
    std::cout << "d_num_neighbors: " << d_num_neighbors.size() << std::endl;
    std::cout << "d_neighbor_list: " << d_neighbor_list.size() << std::endl;
    syncNeighborList();
    kernelUpdateNeighborList<<<dim_grid, dim_block>>>(d_positions_ptr, neighbor_cutoff);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "Particle::updateNeighborList: Error in kernelUpdateNeighborList: " << hipGetErrorString(err) << std::endl;
        exit(EXIT_FAILURE);
    }
    hipDeviceSynchronize();
    max_neighbors = thrust::reduce(d_num_neighbors.begin(), d_num_neighbors.end(), -1L, thrust::maximum<long>());
    std::cout << "max_neighbors: " << max_neighbors << std::endl;
    syncNeighborList();
    std::cout << "max_neighbors_allocated: " << max_neighbors_allocated << std::endl;
    if (max_neighbors > max_neighbors_allocated) {
        max_neighbors_allocated = std::pow(2, std::ceil(std::log2(max_neighbors)));
        d_neighbor_list.resize(n_particles * max_neighbors_allocated);
        std::cout << "d_neighbor_list: " << d_neighbor_list.size() << std::endl;
        thrust::fill(d_neighbor_list.begin(), d_neighbor_list.end(), -1L);
        syncNeighborList();
        kernelUpdateNeighborList<<<dim_grid, dim_block>>>(d_positions_ptr, neighbor_cutoff);
        hipDeviceSynchronize();
    }
}

void Particle::checkForNeighborUpdate() {
    double tolerance = 3.0;
    double max_displacement = getMaxDisplacement();
    if (tolerance * max_displacement > neighbor_cutoff) {
        updateNeighborList();
        thrust::copy(d_positions.begin(), d_positions.end(), d_last_positions.begin());
    }
}