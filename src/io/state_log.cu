#include "hip/hip_runtime.h"
#include "../../include/io/state_log.h"
#include "../../include/io/io_utils.h"
#include <thrust/host_vector.h>

StateLog::StateLog(ConfigDict config, Orchestrator& orchestrator, const std::string& root, const std::string& indexed_file_prefix, const std::string& extension, bool is_restart)
    : BaseLogGroup(config, orchestrator), root(root), indexed_file_prefix(indexed_file_prefix), extension(extension), is_restart(is_restart) {
    this->parallel = true;
}

StateLog::~StateLog() {
}

// need a key: value pair for each reorder array

void StateLog::gather_data(long step) {
    if (orchestrator.arrays_need_reordering) {
        reorder_index_data = orchestrator.get_reorder_index_data();
    }
    for (const auto& name : log_names) {
        ArrayData array_data = orchestrator.get_array_data(name);
        if (array_data.name != "NULL") {
            gathered_data[name] = array_data;
        }
    }
}

void StateLog::write_values(const std::filesystem::path& root, long step) {
    for (auto& [name, array_data] : gathered_data) {
        std::filesystem::path file_path = root / (name + extension);
        if (orchestrator.arrays_need_reordering && reorder_index_data.find(array_data.index_array_name) != reorder_index_data.end()) {
            reorder_array(array_data, reorder_index_data[array_data.index_array_name]);
        }
        write_array_data_to_file(file_path.string(), array_data, precision);
    }
    if (is_restart) {
        // write the step number to a file
        std::filesystem::path step_file_path = root / ("step" + extension);
        std::ofstream step_file(step_file_path.string());
        step_file << step;
        step_file.close();
    }
}

void StateLog::log(long step) {
    std::filesystem::path timestep_root_path;
    if (is_restart) {
        timestep_root_path = std::filesystem::path(root) / "restart";
        make_dir(timestep_root_path.string(), true);
    }
    else {
        timestep_root_path = std::filesystem::path(root) / (indexed_file_prefix + std::to_string(step));
        make_dir(timestep_root_path.string(), true);
    }
    write_values(timestep_root_path, step);
    gathered_data.clear();
    reorder_index_data.clear();
}

void StateLog::write_state_to_path(const std::filesystem::path& path) {
    write_values(path, 0);
}

std::unique_ptr<BaseLogGroup> StateLog::snapshot() const {
    // The default copy constructor will copy the internal state (gathered_data, etc.)
    return std::make_unique<StateLog>(*this);
}