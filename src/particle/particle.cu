#include "hip/hip_runtime.h"
#include "../../include/constants.h"
#include "../../include/functors.h"
#include "../../include/particle/particle.h"
#include "../../include/kernels/kernels.cuh"
#include "../../include/particle/config.h"
#include <iostream>
#include <string>
#include <vector>
#include <cmath>
#include <random>
#include <time.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/fill.h>
#include <thrust/copy.h>
#include <thrust/transform.h>
#include <thrust/transform_reduce.h>
#include <thrust/inner_product.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/permutation_iterator.h>
#include <thrust/functional.h>

Particle::Particle() {
}

Particle::~Particle() {
    clearDynamicVariables();
    clearGeometricVariables();
}

void Particle::initializeFromConfig(const BaseParticleConfig& config) {
}

// ----------------------------------------------------------------------
// ----------------------- Template Methods -----------------------------
// ----------------------------------------------------------------------

std::unordered_map<std::string, std::any> Particle::getArrayMap() {
    std::unordered_map<std::string, std::any> array_map;
    array_map["d_positions"]          = &d_positions;
    array_map["d_last_positions"]     = &d_last_positions;
    array_map["d_displacements"]      = &d_displacements;
    array_map["d_velocities"]         = &d_velocities;
    array_map["d_forces"]             = &d_forces;
    array_map["d_radii"]              = &d_radii;
    array_map["d_masses"]             = &d_masses;
    array_map["d_potential_energy"]   = &d_potential_energy;
    array_map["d_kinetic_energy"]     = &d_kinetic_energy;
    array_map["d_neighbor_list"]      = &d_neighbor_list;
    array_map["d_num_neighbors"]      = &d_num_neighbors;
    array_map["d_cell_index"]         = &d_cell_index;
    array_map["d_sorted_cell_index"]  = &d_sorted_cell_index;
    array_map["d_particle_index"]     = &d_particle_index;
    array_map["d_cell_start"]         = &d_cell_start;
    return array_map;
}

std::string Particle::getArrayType(const std::string& array_name) {
    std::unordered_map<std::string, std::string> array_type_map;
    array_type_map["d_positions"]          = "double";
    array_type_map["d_last_positions"]     = "double";
    array_type_map["d_displacements"]      = "double";
    array_type_map["d_velocities"]         = "double";
    array_type_map["d_forces"]             = "double";
    array_type_map["d_radii"]              = "double";
    array_type_map["d_masses"]             = "double";
    array_type_map["d_potential_energy"]   = "double";
    array_type_map["d_kinetic_energy"]     = "double";
    array_type_map["d_box_size"]           = "double";
    array_type_map["d_neighbor_list"]      = "long";
    array_type_map["d_num_neighbors"]      = "long";
    array_type_map["d_cell_index"]         = "long";
    array_type_map["d_sorted_cell_index"]  = "long";
    array_type_map["d_particle_index"]     = "long";
    array_type_map["d_cell_start"]         = "long";
    return array_type_map[array_name];
}

// ----------------------------------------------------------------------
// -------------------- Universally Defined Methods ---------------------
// ----------------------------------------------------------------------

void Particle::setNeighborListUpdateMethod(std::string method_name) {
    if (method_name == "cell") {
        std::cout << "Particle::setNeighborListUpdateMethod: Setting neighbor list update method to cell" << std::endl;
        this->updateNeighborListPtr = &Particle::updateCellNeighborList;
    } else if (method_name == "verlet") {
        std::cout << "Particle::setNeighborListUpdateMethod: Setting neighbor list update method to verlet" << std::endl;
        this->updateNeighborListPtr = &Particle::updateNeighborList;
    } else if (method_name == "none") {
        std::cout << "Particle::setNeighborListUpdateMethod: Setting neighbor list update method to none" << std::endl;
        throw std::invalid_argument("Particle::setNeighborListUpdateMethod: 'none' neighbor list update method not implemented: " + method_name);
    } else {
        throw std::invalid_argument("Particle::setNeighborListUpdateMethod: Invalid method name: " + method_name);
    }
}

void Particle::setSeed(long seed) {
    if (seed == -1) {
        seed = time(0);
    }
    this->seed = seed;
    srand(seed);
}

void Particle::setNumParticles(long n_particles) {
    this->n_particles = n_particles;
    syncNumParticles();
}

void Particle::syncNumParticles() {
    hipError_t cuda_err = hipMemcpyToSymbol(HIP_SYMBOL(d_n_particles), &n_particles, sizeof(long));
    if (cuda_err != hipSuccess) {
        std::cerr << "Particle::syncNumParticles: Error copying n_particles to device: " << hipGetErrorString(cuda_err) << std::endl;
        exit(EXIT_FAILURE);
    }
}

void Particle::setDegreesOfFreedom() {
    this->n_dof = n_particles * N_DIM;
}

void Particle::setNumVertices(long n_vertices) {
    this->n_vertices = n_vertices;
    syncNumVertices();
}

void Particle::syncNumVertices() {
    hipError_t cuda_err = hipMemcpyToSymbol(HIP_SYMBOL(d_n_vertices), &n_vertices, sizeof(long));
    if (cuda_err != hipSuccess) {
        std::cerr << "Particle::syncNumVertices: Error copying n_vertices to device: " << hipGetErrorString(cuda_err) << std::endl;
        exit(EXIT_FAILURE);
    }
}

void Particle::setParticleCounts(long n_particles, long n_vertices) {
    setNumParticles(n_particles);
    setNumVertices(n_vertices);
    setDegreesOfFreedom();
    initDynamicVariables();
    initGeometricVariables();
}

void Particle::setKernelDimensions(long dim_block) {
    int maxThreadsPerBlock;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    maxThreadsPerBlock = deviceProp.maxThreadsPerBlock;
    std::cout << "CUDA Info: Particle::setKernelDimensions: Max threads per block: " << maxThreadsPerBlock << std::endl;
    if (dim_block > maxThreadsPerBlock) {
        std::cout << "WARNING: Particle::setKernelDimensions: dim_block exceeds maxThreadsPerBlock, adjusting to maxThreadsPerBlock" << std::endl;
        dim_block = maxThreadsPerBlock;
    }
    this->dim_block = dim_block;
    // Implement some particle-specific logic to define the grid dimensions
    // Then, sync
    std::cout << "WARNING: Particle::setKernelDimensions: Not Implemented" << std::endl;
    syncKernelDimensions();
}

void Particle::syncKernelDimensions() {
    hipError_t cuda_err;
    cuda_err = hipMemcpyToSymbol(HIP_SYMBOL(d_dim_block), &dim_block, sizeof(long));
    if (cuda_err != hipSuccess) {
        std::cerr << "Particle::syncKernelDimensions: Error copying dim_block to device: " << hipGetErrorString(cuda_err) << std::endl;
        exit(EXIT_FAILURE);
    }
    cuda_err = hipMemcpyToSymbol(HIP_SYMBOL(d_dim_grid), &dim_grid, sizeof(long));
    if (cuda_err != hipSuccess) {
        std::cerr << "Particle::syncKernelDimensions: Error copying dim_grid to device: " << hipGetErrorString(cuda_err) << std::endl;
        exit(EXIT_FAILURE);
    }
    cuda_err = hipMemcpyToSymbol(HIP_SYMBOL(d_dim_vertex_grid), &dim_vertex_grid, sizeof(long));
    if (cuda_err != hipSuccess) {
        std::cerr << "Particle::syncKernelDimensions: Error copying dim_vertex_grid to device: " << hipGetErrorString(cuda_err) << std::endl;
        exit(EXIT_FAILURE);
    }
}

void Particle::initDynamicVariables() {
    // Resize the device vectors
    d_positions.resize(n_particles * N_DIM);
    d_last_positions.resize(n_particles * N_DIM);
    d_displacements.resize(n_particles * N_DIM);
    d_velocities.resize(n_particles * N_DIM);
    d_forces.resize(n_particles * N_DIM);
    d_radii.resize(n_particles);
    d_masses.resize(n_particles);
    d_potential_energy.resize(n_particles);
    d_kinetic_energy.resize(n_particles);
    d_neighbor_list.resize(n_particles);
    d_num_neighbors.resize(n_particles);

    thrust::fill(d_positions.begin(), d_positions.end(), 0.0);
    thrust::fill(d_last_positions.begin(), d_last_positions.end(), 0.0);
    thrust::fill(d_displacements.begin(), d_displacements.end(), 0.0);
    thrust::fill(d_velocities.begin(), d_velocities.end(), 0.0);
    thrust::fill(d_forces.begin(), d_forces.end(), 0.0);
    thrust::fill(d_radii.begin(), d_radii.end(), 0.0);
    thrust::fill(d_masses.begin(), d_masses.end(), 0.0);
    thrust::fill(d_potential_energy.begin(), d_potential_energy.end(), 0.0);
    thrust::fill(d_kinetic_energy.begin(), d_kinetic_energy.end(), 0.0);

    // max_neighbors = 0;
    // max_neighbors_allocated = 0;
    thrust::fill(d_neighbor_list.begin(), d_neighbor_list.end(), -1L);
    thrust::fill(d_num_neighbors.begin(), d_num_neighbors.end(), max_neighbors);


    // Cast the raw pointers
    d_positions_ptr = thrust::raw_pointer_cast(&d_positions[0]);
    d_last_positions_ptr = thrust::raw_pointer_cast(&d_last_positions[0]);
    d_displacements_ptr = thrust::raw_pointer_cast(&d_displacements[0]);
    d_velocities_ptr = thrust::raw_pointer_cast(&d_velocities[0]);
    d_forces_ptr = thrust::raw_pointer_cast(&d_forces[0]);
    d_radii_ptr = thrust::raw_pointer_cast(&d_radii[0]);
    d_masses_ptr = thrust::raw_pointer_cast(&d_masses[0]);
    d_potential_energy_ptr = thrust::raw_pointer_cast(&d_potential_energy[0]);
    d_kinetic_energy_ptr = thrust::raw_pointer_cast(&d_kinetic_energy[0]);
}

void Particle::clearDynamicVariables() {
    // Clear the device vectors
    d_positions.clear();
    d_last_positions.clear();
    d_displacements.clear();
    d_velocities.clear();
    d_forces.clear();
    d_radii.clear();
    d_masses.clear();
    d_potential_energy.clear();
    d_kinetic_energy.clear();
    d_neighbor_list.clear();
    d_num_neighbors.clear();
    d_cell_index.clear();
    d_sorted_cell_index.clear();
    d_particle_index.clear();
    d_cell_start.clear();

    // Clear the pointers
    d_positions_ptr = nullptr;
    d_last_positions_ptr = nullptr;
    d_displacements_ptr = nullptr;
    d_velocities_ptr = nullptr;
    d_forces_ptr = nullptr;
    d_radii_ptr = nullptr;
    d_masses_ptr = nullptr;
    d_potential_energy_ptr = nullptr;
    d_kinetic_energy_ptr = nullptr;
    d_cell_index_ptr = nullptr;
    d_sorted_cell_index_ptr = nullptr;
    d_particle_index_ptr = nullptr;
    d_cell_start_ptr = nullptr;
}

void Particle::setBoxSize(const thrust::host_vector<double>& box_size) {
    if (box_size.size() != N_DIM) {
        throw std::invalid_argument("Particle::setBoxSize: Error box_size (" + std::to_string(box_size.size()) + ")" + " != " + std::to_string(N_DIM) + " elements");
    }
    hipError_t cuda_err = hipMemcpyToSymbol(HIP_SYMBOL(d_box_size), box_size.data(), sizeof(double) * N_DIM);
    if (cuda_err != hipSuccess) {
        std::cerr << "Particle::setBoxSize: Error copying box size to device: " << hipGetErrorString(cuda_err) << std::endl;
        exit(EXIT_FAILURE);
    }
}

thrust::host_vector<double> Particle::getBoxSize() {
    thrust::host_vector<double> box_size(N_DIM);
    hipError_t cuda_err = hipMemcpyFromSymbol(&box_size[0], HIP_SYMBOL(d_box_size), sizeof(double) * N_DIM);
    if (cuda_err != hipSuccess) {
        std::cerr << "Particle::getBoxSize: Error copying box size to host: " << hipGetErrorString(cuda_err) << std::endl;
        exit(EXIT_FAILURE);
    }
    return box_size;
}

void Particle::syncNeighborList() {
    hipError_t cuda_err = hipMemcpyToSymbol(HIP_SYMBOL(d_max_neighbors_allocated), &this->max_neighbors_allocated, sizeof(this->max_neighbors_allocated));
    if (cuda_err != hipSuccess) {
        std::cerr << "Particle::syncNeighborList: Error copying max_neighbors_allocated to device: " << hipGetErrorString(cuda_err) << std::endl;
        exit(EXIT_FAILURE);
    }
    long* neighbor_list_ptr = thrust::raw_pointer_cast(&d_neighbor_list[0]);
    cuda_err = hipMemcpyToSymbol(HIP_SYMBOL(d_neighbor_list_ptr), &neighbor_list_ptr, sizeof(neighbor_list_ptr));
    if (cuda_err != hipSuccess) {
        std::cerr << "Particle::syncNeighborList: Error copying d_neighbor_list_ptr to device: " << hipGetErrorString(cuda_err) << std::endl;
        exit(EXIT_FAILURE);
    }
    long* num_neighbors_ptr = thrust::raw_pointer_cast(&d_num_neighbors[0]);
    cuda_err = hipMemcpyToSymbol(HIP_SYMBOL(d_num_neighbors_ptr), &num_neighbors_ptr, sizeof(num_neighbors_ptr));
    if (cuda_err != hipSuccess) {
        std::cerr << "Particle::syncNeighborList: Error copying d_num_neighbors_ptr to device: " << hipGetErrorString(cuda_err) << std::endl;
        exit(EXIT_FAILURE);
    }
}

void Particle::setEnergyScale(double e, std::string which) {
    hipError_t cuda_err;
    if (which == "c") {
        e_c = e;
        cuda_err = hipMemcpyToSymbol(HIP_SYMBOL(d_e_c), &e_c, sizeof(double));
        if (cuda_err != hipSuccess) {
            std::cerr << "Particle::setEnergyScale: Error copying e_c to device: " << hipGetErrorString(cuda_err) << std::endl;
            exit(EXIT_FAILURE);
        }
    } else if (which == "a") {
        e_a = e;
        cuda_err = hipMemcpyToSymbol(HIP_SYMBOL(d_e_a), &e_a, sizeof(double));
        if (cuda_err != hipSuccess) {
            std::cerr << "Particle::setEnergyScale: Error copying e_a to device: " << hipGetErrorString(cuda_err) << std::endl;
            exit(EXIT_FAILURE);
        }
    } else if (which == "b") {
        e_b = e;
        cuda_err = hipMemcpyToSymbol(HIP_SYMBOL(d_e_b), &e_b, sizeof(double));
        if (cuda_err != hipSuccess) {
            std::cerr << "Particle::setEnergyScale: Error copying e_b to device: " << hipGetErrorString(cuda_err) << std::endl;
            exit(EXIT_FAILURE);
        }
    } else if (which == "l") {
        e_l = e;
        cuda_err = hipMemcpyToSymbol(HIP_SYMBOL(d_e_l), &e_l, sizeof(double));
        if (cuda_err != hipSuccess) {
            std::cerr << "Particle::setEnergyScale: Error copying e_l to device: " << hipGetErrorString(cuda_err) << std::endl;
            exit(EXIT_FAILURE);
        }
    } else {
        throw std::invalid_argument("Particle::setEnergyScale: which must be 'c', 'a', 'b', or 'l', not " + which);
    }
}

double Particle::getEnergyScale(std::string which) {
    if (which == "c") {
        return e_c;
    } else if (which == "a") {
        return e_a;
    } else if (which == "b") {
        return e_b;
    } else if (which == "l") {
        return e_l;
    } else {
        throw std::invalid_argument("Particle::getEnergyScale: which must be 'c', 'a', 'b', or 'l', not " + which);
    }
}

void Particle::setAllEnergyScales(double e_c, double e_a, double e_b, double e_l) {
    setEnergyScale(e_c, "c");
    setEnergyScale(e_a, "a");
    setEnergyScale(e_b, "b");
    setEnergyScale(e_l, "l");
}

void Particle::setExponent(double n, std::string which) {
    hipError_t cuda_err;
    if (which == "c") {
        n_c = n;
        cuda_err = hipMemcpyToSymbol(HIP_SYMBOL(d_n_c), &n_c, sizeof(double));
        if (cuda_err != hipSuccess) {
            std::cerr << "Particle::setExponent: Error copying n_c to device: " << hipGetErrorString(cuda_err) << std::endl;
            exit(EXIT_FAILURE);
        }
    } else if (which == "a") {
        n_a = n;
        cuda_err = hipMemcpyToSymbol(HIP_SYMBOL(d_n_a), &n_a, sizeof(double));
        if (cuda_err != hipSuccess) {
            std::cerr << "Particle::setExponent: Error copying n_a to device: " << hipGetErrorString(cuda_err) << std::endl;
            exit(EXIT_FAILURE);
        }
    } else if (which == "b") {
        n_b = n;
        cuda_err = hipMemcpyToSymbol(HIP_SYMBOL(d_n_b), &n_b, sizeof(double));
        if (cuda_err != hipSuccess) {
            std::cerr << "Particle::setExponent: Error copying n_b to device: " << hipGetErrorString(cuda_err) << std::endl;
            exit(EXIT_FAILURE);
        }
    } else if (which == "l") {
        n_l = n;
        cuda_err = hipMemcpyToSymbol(HIP_SYMBOL(d_n_l), &n_l, sizeof(double));
        if (cuda_err != hipSuccess) {
            std::cerr << "Particle::setExponent: Error copying n_l to device: " << hipGetErrorString(cuda_err) << std::endl;
            exit(EXIT_FAILURE);
        }
    } else {
        throw std::invalid_argument("Particle::setExponent: which must be 'c', 'a', 'b', or 'l', not " + which);
    }
}

void Particle::setAllExponents(double n_c, double n_a, double n_b, double n_l) {
    setExponent(n_c, "c");
    setExponent(n_a, "a");
    setExponent(n_b, "b");
    setExponent(n_l, "l");
}

double Particle::getExponent(std::string which) {
    if (which == "c") {
        return n_c;
    } else if (which == "a") {
        return n_a;
    } else if (which == "b") {
        return n_b;
    } else if (which == "l") {
        return n_l;
    } else {
        throw std::invalid_argument("Particle::getExponent: which must be 'c', 'a', 'b', or 'l', not " + which);
    }
}

void Particle::initializeBox(double packing_fraction) {
    // set the box size to an arbitrary initial value
    double side_length = 1.0;
    thrust::host_vector<double> box_size(N_DIM, side_length);
    setBoxSize(box_size);
    // then rescale the box size to the desired packing fraction
    scaleToPackingFraction(packing_fraction);
}

void Particle::setRandomUniform(thrust::device_vector<double>& values, double min, double max) {
    thrust::counting_iterator<long> index_sequence_begin(seed);
    thrust::transform(index_sequence_begin, index_sequence_begin + values.size(), values.begin(), RandomUniform(min, max, seed));
}

void Particle::setRandomNormal(thrust::device_vector<double>& values, double mean, double stddev) {
    std::cout << "Set: This does not work yet" << std::endl;
    thrust::counting_iterator<long> index_sequence_begin(seed);
    thrust::transform(index_sequence_begin, index_sequence_begin + values.size(), values.begin(), RandomNormal(mean, stddev, seed));
}

void Particle::setRandomPositions() {
    thrust::host_vector<double> box_size = getBoxSize();
    setRandomUniform(d_positions, 0.0, box_size[0]);
}

void Particle::removeMeanVelocities() {
    std::cout << "Remove: This does not work yet" << std::endl;
    // kernelRemoveMeanVelocities<<<1, N_DIM>>>(d_velocities_ptr);
    // hipDeviceSynchronize();
}

void Particle::scaleVelocitiesToTemperature(double temperature) {
    double current_temp = calculateTemperature();
    thrust::transform(d_velocities.begin(), d_velocities.end(), thrust::make_constant_iterator(std::sqrt(temperature / current_temp)), d_velocities.begin(), thrust::multiplies<double>());
}

void Particle::setRandomVelocities(double temperature) {
    setRandomNormal(d_velocities, 0.0, std::sqrt(temperature));
    removeMeanVelocities();
    scaleVelocitiesToTemperature(temperature);
    // thrust::fill(d_velocities.begin(), d_velocities.end(), 0.0);
}

double Particle::getDiameter(std::string which) {
    if (which == "min") {
        return 2.0 * *thrust::min_element(d_radii.begin(), d_radii.end());
    } else if (which == "max") {
        return 2.0 * *thrust::max_element(d_radii.begin(), d_radii.end());
    } else if (which == "mean") {
        return 2.0 * thrust::reduce(d_radii.begin(), d_radii.end()) / d_radii.size();
    } else {
        throw std::invalid_argument("Particle::getDiameter: which must be 'min', 'max', or 'mean', not " + which);
    }
}

void Particle::setBiDispersity(double size_ratio, double count_ratio) {
    if (size_ratio < 1.0) {
        throw std::invalid_argument("Particle::setBiDispersity: size_ratio must be > 1.0");
    }
    if (count_ratio < 0.0 || count_ratio > 1.0) {
        throw std::invalid_argument("Particle::setBiDispersity: count_ratio must be < 1.0 and > 0.0");
    }
    thrust::host_vector<double> radii(n_particles);
    long n_large = static_cast<long>(n_particles * count_ratio);
    double diam_large = size_ratio;
    double diam_small = 1.0;
    for (long i = 0; i < n_large; i++) {
        radii[i] = diam_large / 2.0;
    }
    for (long i = n_large; i < n_particles; i++) {
        radii[i] = diam_small / 2.0;
    }
    setArray("d_radii", radii);
}

double Particle::getBoxArea() {
    thrust::host_vector<double> box_size = getBoxSize();
    return thrust::reduce(box_size.begin(), box_size.end(), 1.0, thrust::multiplies<double>());
}

double Particle::getPackingFraction() {
    double box_area = getBoxArea();
    double area = getArea();
    return area / box_area;
}

double Particle::getDensity() {
    return getPackingFraction() - getOverlapFraction();
}

void Particle::scaleToPackingFraction(double packing_fraction) {
    double new_side_length = std::pow(getArea() / packing_fraction, 1.0 / N_DIM);
    double side_length = std::pow(getBoxArea(), 1.0 / N_DIM);
    scalePositions(new_side_length / side_length);
    setBoxSize(thrust::host_vector<double>(N_DIM, new_side_length));
}

double Particle::totalKineticEnergy() const {
    return thrust::reduce(d_kinetic_energy.begin(), d_kinetic_energy.end(), 0.0, thrust::plus<double>());
}

double Particle::totalPotentialEnergy() const {
    return thrust::reduce(d_potential_energy.begin(), d_potential_energy.end(), 0.0, thrust::plus<double>());
}

double Particle::totalEnergy() const {
    return totalKineticEnergy() + totalPotentialEnergy();
}

void Particle::scalePositions(double scale_factor) {
    thrust::transform(d_positions.begin(), d_positions.end(), thrust::make_constant_iterator(scale_factor), d_positions.begin(), thrust::multiplies<double>());
}

void Particle::updatePositions(double dt) {
    kernelUpdatePositions<<<dim_grid, dim_block>>>(d_positions_ptr, d_last_positions_ptr, d_displacements_ptr, d_velocities_ptr, dt);
}

void Particle::updateVelocities(double dt) {
    kernelUpdateVelocities<<<dim_grid, dim_block>>>(d_velocities_ptr, d_forces_ptr, d_masses_ptr, dt);
}

double Particle::getMaxDisplacement() {
    return thrust::reduce(d_displacements.begin(), d_displacements.end(), 0.0, thrust::maximum<double>());
}

void Particle::updateNeighborList() {
    thrust::fill(d_neighbor_list.begin(), d_neighbor_list.end(), -1L);
    kernelUpdateNeighborList<<<dim_grid, dim_block>>>(d_positions_ptr, neighbor_cutoff);
    max_neighbors = thrust::reduce(d_num_neighbors.begin(), d_num_neighbors.end(), -1L, thrust::maximum<long>());
    if (max_neighbors > max_neighbors_allocated) {
        max_neighbors_allocated = std::pow(2, std::ceil(std::log2(max_neighbors)));
        std::cout << "Particle::updateNeighborList: Resizing neighbor list to " << max_neighbors_allocated << std::endl;
        d_neighbor_list.resize(n_particles * max_neighbors_allocated);
        thrust::fill(d_neighbor_list.begin(), d_neighbor_list.end(), -1L);
        syncNeighborList();
        kernelUpdateNeighborList<<<dim_grid, dim_block>>>(d_positions_ptr, neighbor_cutoff);
    }
}

void Particle::checkForNeighborUpdate() {
    double tolerance = 3.0;
    double max_displacement = getMaxDisplacement();
    if (tolerance * max_displacement > neighbor_displacement) {
        (this->*updateNeighborListPtr)();
        thrust::copy(d_positions.begin(), d_positions.end(), d_last_positions.begin());
        thrust::fill(d_displacements.begin(), d_displacements.end(), 0.0);
    }
}

void Particle::initializeNeighborList() {
    d_neighbor_list.resize(n_particles * max_neighbors_allocated);
    d_num_neighbors.resize(n_particles);
    thrust::fill(d_num_neighbors.begin(), d_num_neighbors.end(), 0L);
    thrust::fill(d_neighbor_list.begin(), d_neighbor_list.end(), -1L);
    syncNeighborList();
    updateNeighborList();
}

void Particle::setNeighborCutoff(double neighbor_cutoff_multiplier, double neighbor_displacement_multiplier) {
    this->neighbor_cutoff = neighbor_cutoff_multiplier * getDiameter("max");
    this->neighbor_displacement = neighbor_displacement_multiplier * neighbor_cutoff;
    this->max_neighbors_allocated = 4;

    thrust::host_vector<double> box_size = getBoxSize();
    std::cout << "Particle::setNeighborCutoff: Neighbor cutoff set to " << neighbor_cutoff << " and neighbor displacement set to " << neighbor_displacement << " box length: " << box_size[0] << std::endl;
}

void Particle::printNeighborList() {
    thrust::host_vector<long> neighbor_list = getArray<long>("d_neighbor_list");
    thrust::host_vector<long> num_neighbors = getArray<long>("d_num_neighbors");
    for (long i = 0; i < n_particles; i++) {
        std::cout << "Particle " << i << " has " << num_neighbors[i] << " neighbors." << std::endl;
        for (long j = 0; j < num_neighbors[i]; j++) {
            std::cout << "\t\tNeighbor " << j << " of particle " << i << " is " << neighbor_list[i * max_neighbors + j] << std::endl;
        }
    }
}

void Particle::setCellSize(double cell_size_multiplier) {
    long min_num_cells_dim = 4;  // if there are fewer than 4 cells in one axis, the cell list probably wont work
    double trial_cell_size = cell_size_multiplier * getDiameter("max");
    thrust::host_vector<double> box_size = getBoxSize();
    n_cells_dim = static_cast<long>(std::floor(box_size[0] / trial_cell_size));
    n_cells = n_cells_dim * n_cells_dim;
    if (n_cells_dim < min_num_cells_dim) {
        throw std::runtime_error("Particle::setCellSize: fewer than " + std::to_string(min_num_cells_dim) + " cells in one dimension");
    }
    cell_size = box_size[0] / n_cells_dim;
    std::cout << "Particle::setCellSize: Cell size set to " << cell_size << std::endl;
    syncCellList();
}

void Particle::initializeCellList() {
    d_cell_index.resize(n_particles);
    d_sorted_cell_index.resize(n_particles);
    d_particle_index.resize(n_particles);
    d_cell_start.resize(n_cells + 1);

    thrust::fill(d_cell_index.begin(), d_cell_index.end(), -1L);
    thrust::fill(d_sorted_cell_index.begin(), d_sorted_cell_index.end(), -1L);
    thrust::fill(d_particle_index.begin(), d_particle_index.end(), -1L);
    thrust::fill(d_cell_start.begin(), d_cell_start.end(), -1L);

    d_cell_index_ptr = thrust::raw_pointer_cast(d_cell_index.data());
    d_sorted_cell_index_ptr = thrust::raw_pointer_cast(d_sorted_cell_index.data());
    d_particle_index_ptr = thrust::raw_pointer_cast(d_particle_index.data());
    d_cell_start_ptr = thrust::raw_pointer_cast(d_cell_start.data());
}

void Particle::syncCellList() {
    hipError_t cuda_err = hipMemcpyToSymbol(HIP_SYMBOL(d_n_cells), &n_cells, sizeof(long));
    if (cuda_err != hipSuccess) {
        std::cerr << "Particle::syncCellList: Error copying n_cells to device: " << hipGetErrorString(cuda_err) << std::endl;
        exit(EXIT_FAILURE);
    }
    cuda_err = hipMemcpyToSymbol(HIP_SYMBOL(d_n_cells_dim), &n_cells_dim, sizeof(long));
    if (cuda_err != hipSuccess) {
        std::cerr << "Particle::syncCellList: Error copying n_cells_dim to device: " << hipGetErrorString(cuda_err) << std::endl;
        exit(EXIT_FAILURE);
    }
    cuda_err = hipMemcpyToSymbol(HIP_SYMBOL(d_cell_size), &cell_size, sizeof(double));
    if (cuda_err != hipSuccess) {
        std::cerr << "Particle::syncCellList: Error copying cell_size to device: " << hipGetErrorString(cuda_err) << std::endl;
        exit(EXIT_FAILURE);
    }
}

void Particle::updateCellList() {
    d_cell_start[n_cells] = n_particles;
    kernelGetCellIndexForParticle<<<dim_grid, dim_block>>>(d_positions_ptr, d_cell_index_ptr, d_sorted_cell_index_ptr, d_particle_index_ptr);
    thrust::sort_by_key(d_sorted_cell_index.begin(), d_sorted_cell_index.end(), d_particle_index.begin());
    // TODO: this is a kernel over cells - could probably be parallelized better
    long width_offset = 2;
    long width = n_particles / n_cells;
    kernelGetFirstParticleIndexForCell<<<dim_grid, dim_block>>>(d_sorted_cell_index_ptr, d_cell_start_ptr, width_offset, width);
}

void Particle::updateCellNeighborList() {
    updateCellList();
    thrust::fill(d_neighbor_list.begin(), d_neighbor_list.end(), -1L);
    kernelUpdateCellNeighborList<<<dim_grid, dim_block>>>(d_positions_ptr, neighbor_cutoff, d_cell_index_ptr, d_particle_index_ptr, d_cell_start_ptr);
    max_neighbors = thrust::reduce(d_num_neighbors.begin(), d_num_neighbors.end(), -1L, thrust::maximum<long>());
    if (max_neighbors > max_neighbors_allocated) {
        max_neighbors_allocated = std::pow(2, std::ceil(std::log2(max_neighbors)));
        std::cout << "Particle::updateCellNeighborList: Resizing neighbor list to " << max_neighbors_allocated << std::endl;
        d_neighbor_list.resize(n_particles * max_neighbors_allocated);
        thrust::fill(d_neighbor_list.begin(), d_neighbor_list.end(), -1L);
        syncNeighborList();
        kernelUpdateCellNeighborList<<<dim_grid, dim_block>>>(d_positions_ptr, neighbor_cutoff, d_cell_index_ptr, d_particle_index_ptr, d_cell_start_ptr);
    }
}

void Particle::zeroForceAndPotentialEnergy() {
    thrust::fill(d_forces.begin(), d_forces.end(), 0.0);
    thrust::fill(d_potential_energy.begin(), d_potential_energy.end(), 0.0);
}

double Particle::calculateTemperature() {
    calculateKineticEnergy();
    return totalKineticEnergy() * 2.0 / n_dof;
}

double Particle::getTimeUnit() {
    double average_mass = thrust::reduce(d_masses.begin(), d_masses.end()) / n_particles;
    return getDiameter("min") * std::sqrt(average_mass / getEnergyScale("c"));
}

void Particle::setMass(double mass) {
    thrust::fill(d_masses.begin(), d_masses.end(), mass);
}