#include "hip/hip_runtime.h"
#include "../../include/routines/compression.h"

void jam_adam(Particle& particle, Adam& adam, IOManager& io_manager, long num_compression_steps, long num_adam_steps, double avg_pe_target, double avg_pe_diff_target, double packing_fraction_increment, double min_packing_fraction_increment, double max_pe_target) {
    particle.initAdamVariables();
    particle.calculateParticleArea();
    double packing_fraction = particle.getPackingFraction();

    long compression_step = 0;
    double avg_pe_past_jamming = 1e-9;  // marks being above jamming (might be too high)
    double avg_pe = 0.0;
    double dof = static_cast<double>(particle.n_dof);
    double last_avg_pe = 0.0;
    double avg_pe_diff = 0.0;
    long adam_step = 0;
    double sign = 1.0;
    while (compression_step < num_compression_steps && avg_pe < avg_pe_past_jamming) {
        adam_step = 0;
        last_avg_pe = 0.0;
        avg_pe_diff = 0.0;
        while (adam_step < num_adam_steps) {
            adam.minimize(adam_step);
            avg_pe = particle.totalPotentialEnergy() / dof / particle.e_c;
            avg_pe_diff = std::abs(avg_pe - last_avg_pe);
            last_avg_pe = avg_pe;
            if (avg_pe_diff < avg_pe_diff_target || avg_pe < avg_pe_target) {
                break;
            }
            adam_step++;
        }
        if (avg_pe > max_pe_target) {
            sign = -1.0;
            if (packing_fraction_increment > min_packing_fraction_increment) {
                packing_fraction_increment /= 2.0;
            }
        } else if (avg_pe > avg_pe_target) {
            std::cout << "jamming complete" << std::endl;
            break;
        } else {
            sign = 1.0;
        }
        io_manager.log(compression_step);
        particle.scaleToPackingFraction(packing_fraction + packing_fraction_increment * sign);
        packing_fraction = particle.getPackingFraction();
        compression_step++;
    }   
}