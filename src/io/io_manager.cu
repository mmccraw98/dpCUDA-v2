#include "hip/hip_runtime.h"
#include "../../include/io/io_manager.h"
#include <iostream>
#include <filesystem>

IOManager::IOManager(std::vector<LogGroupConfig> log_configs, Particle& particle, Integrator* integrator, std::string root_path, bool overwrite) : particle(particle), integrator(integrator), orchestrator(particle, integrator), root_path(root_path), overwrite(overwrite), log_configs(log_configs) {
    // probably validate root_path if it is not empty

    for (auto& config : log_configs) {

        if (config.group_name == "energy") {
            if (system_dir_path.empty()) {
                init_path(system_dir_path, system_dir_name);
                make_dir(system_dir_path, overwrite);  // may need to change function signature
            }
            std::filesystem::path energy_file_path = system_dir_path / (energy_file_name + energy_file_extension);
            log_groups.push_back(new EnergyLog(config, orchestrator, energy_file_path, overwrite));

        } else if (config.group_name == "console") {
            log_groups.push_back(new ConsoleLog(config, orchestrator));

        } else if (config.group_name == "state") {
            if (trajectory_dir_path.empty()) {
                init_path(trajectory_dir_path, trajectory_dir_name);
                make_dir(trajectory_dir_path, overwrite);  // may need to change function signature
            }
            log_groups.push_back(new StateLog(config, orchestrator, trajectory_dir_path, indexed_file_prefix, state_file_extension));
        
        } else if (config.group_name == "init") {
            if (system_dir_path.empty()) {
                init_path(system_dir_path, system_dir_name);
                make_dir(system_dir_path, overwrite);  // may need to change function signature
            }
            state_log = new StateLog(config, orchestrator, system_dir_path, "", state_file_extension);
            state_log->write_state();

        } else {
            std::cerr << "ERROR: IOManager::IOManager:" << config.group_name << " is not a valid log group name" << std::endl;
        }
    }
}

IOManager::~IOManager() {
    for (auto& log_group : log_groups) {
        delete log_group;
    }
}

void IOManager::init_path(std::filesystem::path& path, const std::string& path_name) {
    if (root_path.empty()) {
        std::cerr << "ERROR: IOManager::init_path:" << path_name << " root_path is empty" << std::endl;
        return;
    }
    path = static_cast<std::filesystem::path>(root_path) / static_cast<std::filesystem::path>(path_name);
}

void IOManager::log(long step) {
    bool log_required = false;
    for (BaseLogGroup* log_group : log_groups) {
        log_group->update_log_status(step);
        if (log_group->should_log) {
            log_required = true;
        }
    }

    if (log_required) {
        orchestrator.init_pre_req_calculation_status();
        for (BaseLogGroup* log_group : log_groups) {
            if (log_group->should_log) {
                log_group->log(step);
            }
        }
    }
}

void IOManager::write_log_configs(std::filesystem::path path) {
    nlohmann::json all_configs_json;
    for (auto& config : log_configs) {
        all_configs_json[config.group_name] = config.to_json();
    }
    write_json_to_file(path / "log_configs.json", all_configs_json);
}

void IOManager::write_particle_config(std::filesystem::path path) {
    write_json_to_file(path / "particle_config.json", particle.config->to_json());
}

void IOManager::write_integrator_config(std::filesystem::path path) {
    write_json_to_file(path / "integrator_config.json", integrator->config.to_json());
}

void IOManager::write_params() {
    if (system_dir_path.empty()) {
        init_path(system_dir_path, system_dir_name);
        make_dir(system_dir_path, overwrite);  // may need to change function signature
    }
    write_log_configs(system_dir_path);
    write_particle_config(system_dir_path);
    if (integrator != nullptr) {
        write_integrator_config(system_dir_path);
    }
    // TODO: write run params
}

// void IOManager::write_state() {
//     if (state_log == nullptr) {
//         std::cerr << "ERROR: IOManager::write_state: state_log is not initialized" << std::endl;
//         return;
//     }
//     state_log->write_state();
// }