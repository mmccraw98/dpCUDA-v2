#include "hip/hip_runtime.h"
#include "../../include/constants.h"
#include "../../include/functors.h"
#include "../../include/particle/particle.h"
#include "../../include/kernels/kernels.cuh"
#include "../../include/particle/config.h"
#include <iostream>
#include <string>
#include <vector>
#include <cmath>
#include <random>
#include <time.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/fill.h>
#include <thrust/copy.h>
#include <thrust/transform.h>
#include <thrust/transform_reduce.h>
#include <thrust/inner_product.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/permutation_iterator.h>
#include <thrust/functional.h>


#include <typeinfo>
template <typename T>
void printType(const T& obj) {
    std::cout << "Type: " << typeid(obj).name() << std::endl;
}

#define CUDA_CHECK(call)                                                    \
    {                                                                       \
        hipError_t err = call;                                             \
        if (err != hipSuccess) {                                           \
            std::cerr << "CUDA error in " << __FILE__ << " at line "        \
                      << __LINE__ << ": " << hipGetErrorString(err) << "\n"; \
            std::exit(EXIT_FAILURE);                                        \
        }                                                                   \
    }

Particle::Particle() {
}

Particle::~Particle() {
    clearDynamicVariables();
    clearGeometricVariables();
    clearNeighborVariables();
}

void Particle::initializeFromConfig(const BaseParticleConfig& config) {
    if (const auto* bidisperse_config = dynamic_cast<const BidisperseParticleConfig*>(&config)) {
        this->config = std::make_unique<BidisperseParticleConfig>(*bidisperse_config);
    } else {
        throw std::runtime_error("ERROR: Disk::initializeFromConfig: Invalid configuration type.");
    }

    this->define_unique_dependencies();

    this->setSeed(config.seed);
    this->setParticleCounts(config.n_particles, 0);
    this->setKernelDimensions(config.particle_dim_block);

    // Dynamic cast to check if config is BidisperseParticleConfig
    if (const auto* bidisperse_config = dynamic_cast<const BidisperseParticleConfig*>(&config)) {
        // Handle bidisperse-specific fields
        this->setBiDispersity(bidisperse_config->size_ratio, bidisperse_config->count_ratio);
    } else {
        throw std::runtime_error("ERROR: Disk::initializeFromConfig: Invalid configuration type.");
    }
    this->initializeBox(config.packing_fraction);

    // TODO: make this a config - position initialization config: zero, random, etc.
    this->setRandomPositions();

    this->setEnergyScale(config.e_c, "c");
    this->setExponent(config.n_c, "c");
    this->setMass(config.mass);

    this->setNeighborMethod(config.neighbor_list_update_method);
    this->setNeighborSize(config.neighbor_cutoff_multiplier, config.neighbor_displacement_multiplier);

    if (this->neighbor_list_update_method == "cell") {
        bool could_set_cell_size = this->setCellSize(config.num_particles_per_cell, config.cell_displacement_multiplier);
        if (!could_set_cell_size) {
            std::cout << "WARNING: Disk::initializeFromConfig: Could not set cell size.  Attempting to use verlet list instead." << std::endl;
            this->setNeighborMethod("verlet");
        }
        bool could_set_neighbor_size = this->setNeighborSize(config.neighbor_cutoff_multiplier, config.neighbor_displacement_multiplier);
        if (!could_set_neighbor_size) {
            std::cerr << "ERROR: Disk::initializeFromConfig: Could not set neighbor size for cell list - neighbor cutoff exceeds box size.  Attempting to use all-to-all instead." << std::endl;
            this->setNeighborMethod("all");
        }
    }
    if (this->neighbor_list_update_method == "verlet") {
        bool could_set_neighbor_size = this->setNeighborSize(config.neighbor_cutoff_multiplier, config.neighbor_displacement_multiplier);
        if (!could_set_neighbor_size) {
            std::cout << "WARNING: Disk::initializeFromConfig: Could not set neighbor size.  Attempting to use all-to-all instead." << std::endl;
            this->setNeighborMethod("all");
        }
    }
    this->initNeighborList();
    this->calculateForces();  // make sure forces are calculated before the integration starts
    // may want to check that the forces are balanced
}

void Particle::setNeighborMethod(std::string method_name) {
    this->using_cell_list = false;
    this->neighbor_list_update_method = method_name;
    if (method_name == "cell") {
        std::cout << "Particle::setNeighborMethod: Setting neighbor list update method to cell" << std::endl;
        this->initNeighborListPtr = &Particle::initCellList;
        this->updateNeighborListPtr = &Particle::updateCellNeighborList;
        this->checkForNeighborUpdatePtr = &Particle::checkForCellListUpdate;
        this->using_cell_list = true;
    } else if (method_name == "verlet") {
        std::cout << "Particle::setNeighborMethod: Setting neighbor list update method to verlet" << std::endl;
        this->initNeighborListPtr = &Particle::initVerletList;
        this->updateNeighborListPtr = &Particle::updateVerletList;
        this->checkForNeighborUpdatePtr = &Particle::checkForVerletListUpdate;
    } else if (method_name == "all") {
        std::cout << "Particle::setNeighborMethod: Setting neighbor list update method to all" << std::endl;
        thrust::host_vector<double> host_box_size = box_size.getData();
        double max_diameter = getDiameter("max");
        double box_diagonal = std::sqrt(host_box_size[0] * host_box_size[0] + host_box_size[1] * host_box_size[1]);
        double neighbor_cutoff_multiplier = 2.0 * box_diagonal / max_diameter;  // set it to be twice the diagonal length so that every particle is included always (2x multiplier is extraneous but harmless)
        setNeighborSize(neighbor_cutoff_multiplier, 0.0);  // the neighbor displacement is unused here
        this->initNeighborListPtr = &Particle::initAllToAllList;
        this->updateNeighborListPtr = &Particle::updateVerletList;
        this->checkForNeighborUpdatePtr = &Particle::checkForAllToAllUpdate;
    } else {
        throw std::invalid_argument("Particle::setNeighborMethod: Invalid method name: " + method_name);
    }
}

void Particle::setSeed(long seed) {
    if (seed == -1) {
        seed = time(0);
    }
    this->seed = seed;
    srand(seed);
}

void Particle::setNumParticles(long n_particles) {
    this->n_particles = n_particles;
    syncNumParticles();
}

void Particle::syncNumParticles() {
    hipError_t cuda_err = hipMemcpyToSymbol(HIP_SYMBOL(d_n_particles), &n_particles, sizeof(long));
    if (cuda_err != hipSuccess) {
        std::cerr << "Particle::syncNumParticles: Error copying n_particles to device: " << hipGetErrorString(cuda_err) << std::endl;
        exit(EXIT_FAILURE);
    }
}

void Particle::setDegreesOfFreedom() {
    this->n_dof = n_particles * N_DIM;
}

void Particle::setNumVertices(long n_vertices) {
    this->n_vertices = n_vertices;
    syncNumVertices();
}

void Particle::syncNumVertices() {
    hipError_t cuda_err = hipMemcpyToSymbol(HIP_SYMBOL(d_n_vertices), &n_vertices, sizeof(long));
    if (cuda_err != hipSuccess) {
        std::cerr << "Particle::syncNumVertices: Error copying n_vertices to device: " << hipGetErrorString(cuda_err) << std::endl;
        exit(EXIT_FAILURE);
    }
}

void Particle::setParticleCounts(long n_particles, long n_vertices) {
    setNumParticles(n_particles);
    setNumVertices(n_vertices);
    setDegreesOfFreedom();
    initDynamicVariables();
    initGeometricVariables();
}

void Particle::setKernelDimensions(long particle_dim_block) {
    int maxThreadsPerBlock;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    maxThreadsPerBlock = deviceProp.maxThreadsPerBlock;
    std::cout << "CUDA Info: Particle::setKernelDimensions: Max threads per block: " << maxThreadsPerBlock << std::endl;
    if (particle_dim_block > maxThreadsPerBlock) {
        std::cout << "WARNING: Particle::setKernelDimensions: particle_dim_block exceeds maxThreadsPerBlock, adjusting to maxThreadsPerBlock" << std::endl;
        particle_dim_block = maxThreadsPerBlock;
    }
    this->particle_dim_block = particle_dim_block;
    // Implement some particle-specific logic to define the grid dimensions
    // Then, sync
    std::cout << "WARNING: Particle::setKernelDimensions: Not Implemented" << std::endl;
    syncKernelDimensions();
}

void Particle::syncKernelDimensions() {
    hipError_t cuda_err;
    cuda_err = hipMemcpyToSymbol(HIP_SYMBOL(d_dim_block), &particle_dim_block, sizeof(long));
    if (cuda_err != hipSuccess) {
        std::cerr << "Particle::syncKernelDimensions: Error copying particle_dim_block to device: " << hipGetErrorString(cuda_err) << std::endl;
        exit(EXIT_FAILURE);
    }
    cuda_err = hipMemcpyToSymbol(HIP_SYMBOL(d_dim_grid), &particle_dim_grid, sizeof(long));
    if (cuda_err != hipSuccess) {
        std::cerr << "Particle::syncKernelDimensions: Error copying particle_dim_grid to device: " << hipGetErrorString(cuda_err) << std::endl;
        exit(EXIT_FAILURE);
    }
    cuda_err = hipMemcpyToSymbol(HIP_SYMBOL(d_dim_vertex_grid), &vertex_dim_grid, sizeof(long));
    if (cuda_err != hipSuccess) {
        std::cerr << "Particle::syncKernelDimensions: Error copying vertex_dim_grid to device: " << hipGetErrorString(cuda_err) << std::endl;
        exit(EXIT_FAILURE);
    }
}

void Particle::initDynamicVariables() {
    // Resize and fill all the device arrays to avoid any potential issues with uninitialized data
    positions.resizeAndFill(n_particles, 0.0, 0.0);
    velocities.resizeAndFill(n_particles, 0.0, 0.0);
    forces.resizeAndFill(n_particles, 0.0, 0.0);
    radii.resizeAndFill(n_particles, 0.0);
    masses.resizeAndFill(n_particles, 0.0);
    kinetic_energy.resizeAndFill(n_particles, 0.0);
    potential_energy.resizeAndFill(n_particles, 0.0);
}

void Particle::clearDynamicVariables() {
    positions.clear();
    velocities.clear();
    forces.clear();
    radii.clear();
    masses.clear();
    kinetic_energy.clear();
    potential_energy.clear();
}

void Particle::clearNeighborVariables() {
    neighbor_list.clear();
    num_neighbors.clear();
    cell_index.clear();
    particle_index.clear();
    static_particle_index.clear();
    cell_start.clear();
    last_neigh_positions.clear();
    last_cell_positions.clear();
    neigh_displacements_sq.clear();
    cell_displacements_sq.clear();
}

void Particle::define_unique_dependencies() {
    for (const auto& pair : calculation_dependencies) {
        unique_dependents.insert(pair.first);
        for (const auto& dependency : pair.second) {
            unique_dependencies.insert(dependency);
        }
    }
    reset_dependency_status();
}

void Particle::reset_dependency_status() {
    dependency_status.clear();
    for (const auto& dependency : unique_dependencies) {
        dependency_status[dependency] = false;
    }
}

void Particle::calculate_dependencies(std::string log_name) {
    for (const auto& dependency : calculation_dependencies[log_name]) {
        if (!dependency_status[dependency]) {
            calculate_dependencies(dependency);  // handle nested dependencies
            handle_calculation_for_single_dependency(dependency);
            dependency_status[dependency] = true;  // once calculated, it doesnt need to be calculated again
        }
    }
}

void Particle::handle_calculation_for_single_dependency(std::string dependency_calculation_name) {
    // logic to calculate the dependency goes here - need one for each value in unique_dependencies
    if (dependency_calculation_name == "calculate_kinetic_energy") {
        calculateKineticEnergy();
    } 
    // fill in the rest here....

    else {
        throw std::invalid_argument("Particle::handle_calculation_for_single_dependency: dependency_calculation_name not found: " + dependency_calculation_name);
    }
}

ArrayData Particle::getArrayData(const std::string& array_name) {
    ArrayData result;
    result.name = array_name;
    if (array_name == "positions") {
        result.type = DataType::Double;
        result.size = positions.size;
        result.data = std::make_pair(positions.getDataX(), positions.getDataY());
        result.index_array_name = "static_particle_index";
    } else if (array_name == "velocities") {
        result.type = DataType::Double;
        result.size = velocities.size;
        result.data = std::make_pair(velocities.getDataX(), velocities.getDataY());
        result.index_array_name = "static_particle_index";
    } else if (array_name == "forces") {
        result.type = DataType::Double;
        result.size = forces.size;
        result.data = std::make_pair(forces.getDataX(), forces.getDataY());
        result.index_array_name = "static_particle_index";
    } else if (array_name == "box_size") {
        result.type = DataType::Double;
        result.size = box_size.size;
        result.data = box_size.getData();
        result.index_array_name = "";
    } else if (array_name == "radii") {
        result.type = DataType::Double;
        result.size = radii.size;
        result.data = radii.getData();
        result.index_array_name = "static_particle_index";
    } else if (array_name == "masses") {
        result.type = DataType::Double;
        result.size = masses.size;
        result.data = masses.getData();
        result.index_array_name = "static_particle_index";
    } else if (array_name == "kinetic_energy") {
        result.type = DataType::Double;
        result.size = kinetic_energy.size;
        result.data = kinetic_energy.getData();
        result.index_array_name = "static_particle_index";
    } else if (array_name == "potential_energy") {
        result.type = DataType::Double;
        result.size = potential_energy.size;
        result.data = potential_energy.getData();
        result.index_array_name = "static_particle_index";
    } else if (array_name == "neighbor_list") {
        result.type = DataType::Long;
        result.size = neighbor_list.size;
        result.data = neighbor_list.getData();
        result.index_array_name = ""; // this is a tricky one to incorporate in the reordering process in a general way
    } else if (array_name == "num_neighbors") {
        result.type = DataType::Long;
        result.size = num_neighbors.size;
        result.data = num_neighbors.getData();
        result.index_array_name = "static_particle_index";
    } else if (array_name == "cell_index") {
        result.type = DataType::Long;
        result.size = cell_index.size;
        result.data = cell_index.getData();
        result.index_array_name = "static_particle_index";
    } else if (array_name == "particle_index") {
        result.type = DataType::Long;
        result.size = particle_index.size;
        result.data = particle_index.getData();
        result.index_array_name = "static_particle_index";
    } else if (array_name == "static_particle_index") {
        result.type = DataType::Long;
        result.size = static_particle_index.size;
        result.data = static_particle_index.getData();
        result.index_array_name = "";
    } else if (array_name == "cell_start") {
        result.type = DataType::Long;
        result.size = cell_start.size;
        result.data = cell_start.getData();
        result.index_array_name = "";
    } else {
        throw std::invalid_argument("Particle::getArrayData: array_name " + array_name + " not found");
    }
    return result;
}

void Particle::setBoxSize(const thrust::host_vector<double>& host_box_size) {  // TODO: work on this
    if (host_box_size.size() != N_DIM) {
        throw std::invalid_argument("Particle::setBoxSize: Error box_size (" + std::to_string(host_box_size.size()) + ")" + " != " + std::to_string(N_DIM) + " elements");
    }
    box_size.resize(N_DIM);
    box_size.setData(host_box_size);
    hipError_t cuda_err = hipMemcpyToSymbol(HIP_SYMBOL(d_box_size), box_size.getData().data(), sizeof(double) * N_DIM);
    if (cuda_err != hipSuccess) {
        std::cerr << "Particle::setBoxSize: Error copying box size to device: " << hipGetErrorString(cuda_err) << std::endl;
        exit(EXIT_FAILURE);  // TODO: make this a function and put it in a cuda module
    }
}

void Particle::syncNeighborList() {
    hipError_t cuda_err = hipMemcpyToSymbol(HIP_SYMBOL(d_max_neighbors_allocated), &this->max_neighbors_allocated, sizeof(this->max_neighbors_allocated));
    if (cuda_err != hipSuccess) {
        std::cerr << "Particle::syncNeighborList: Error copying max_neighbors_allocated to device: " << hipGetErrorString(cuda_err) << std::endl;
        exit(EXIT_FAILURE);  // TODO: make this a function and put it in a cuda module
    }
    cuda_err = hipMemcpyToSymbol(HIP_SYMBOL(d_neighbor_list_ptr), &neighbor_list.d_ptr, sizeof(neighbor_list.d_ptr));
    if (cuda_err != hipSuccess) {
        std::cerr << "Particle::syncNeighborList: Error copying d_neighbor_list_ptr to device: " << hipGetErrorString(cuda_err) << std::endl;
        exit(EXIT_FAILURE);  // TODO: make this a function and put it in a cuda module
    }
    cuda_err = hipMemcpyToSymbol(HIP_SYMBOL(d_num_neighbors_ptr), &num_neighbors.d_ptr, sizeof(num_neighbors.d_ptr));
    if (cuda_err != hipSuccess) {
        std::cerr << "Particle::syncNeighborList: Error copying d_num_neighbors_ptr to device: " << hipGetErrorString(cuda_err) << std::endl;
        exit(EXIT_FAILURE);  // TODO: make this a function and put it in a cuda module
    }
}

void Particle::setEnergyScale(double e, std::string which) {
    hipError_t cuda_err;
    if (which == "c") {
        e_c = e;
        cuda_err = hipMemcpyToSymbol(HIP_SYMBOL(d_e_c), &e_c, sizeof(double));
        if (cuda_err != hipSuccess) {
            std::cerr << "Particle::setEnergyScale: Error copying e_c to device: " << hipGetErrorString(cuda_err) << std::endl;
            exit(EXIT_FAILURE);  // TODO: make this a function and put it in a cuda module
        }
    } else if (which == "a") {
        e_a = e;
        cuda_err = hipMemcpyToSymbol(HIP_SYMBOL(d_e_a), &e_a, sizeof(double));
        if (cuda_err != hipSuccess) {
            std::cerr << "Particle::setEnergyScale: Error copying e_a to device: " << hipGetErrorString(cuda_err) << std::endl;
            exit(EXIT_FAILURE);  // TODO: make this a function and put it in a cuda module
        }
    } else if (which == "b") {
        e_b = e;
        cuda_err = hipMemcpyToSymbol(HIP_SYMBOL(d_e_b), &e_b, sizeof(double));
        if (cuda_err != hipSuccess) {
            std::cerr << "Particle::setEnergyScale: Error copying e_b to device: " << hipGetErrorString(cuda_err) << std::endl;
            exit(EXIT_FAILURE);  // TODO: make this a function and put it in a cuda module
        }
    } else if (which == "l") {
        e_l = e;
        cuda_err = hipMemcpyToSymbol(HIP_SYMBOL(d_e_l), &e_l, sizeof(double));
        if (cuda_err != hipSuccess) {
            std::cerr << "Particle::setEnergyScale: Error copying e_l to device: " << hipGetErrorString(cuda_err) << std::endl;
            exit(EXIT_FAILURE);  // TODO: make this a function and put it in a cuda module
        }
    } else {
        throw std::invalid_argument("Particle::setEnergyScale: which must be 'c', 'a', 'b', or 'l', not " + which);
    }
}

double Particle::getEnergyScale(std::string which) {
    if (which == "c") {
        return e_c;
    } else if (which == "a") {
        return e_a;
    } else if (which == "b") {
        return e_b;
    } else if (which == "l") {
        return e_l;
    } else {
        throw std::invalid_argument("Particle::getEnergyScale: which must be 'c', 'a', 'b', or 'l', not " + which);
    }
}

void Particle::setAllEnergyScales(double e_c, double e_a, double e_b, double e_l) {
    setEnergyScale(e_c, "c");
    setEnergyScale(e_a, "a");
    setEnergyScale(e_b, "b");
    setEnergyScale(e_l, "l");
}

void Particle::setExponent(double n, std::string which) {
    hipError_t cuda_err;
    if (which == "c") {
        n_c = n;
        cuda_err = hipMemcpyToSymbol(HIP_SYMBOL(d_n_c), &n_c, sizeof(double));
        if (cuda_err != hipSuccess) {
            std::cerr << "Particle::setExponent: Error copying n_c to device: " << hipGetErrorString(cuda_err) << std::endl;
            exit(EXIT_FAILURE);  // TODO: make this a function and put it in a cuda module
        }
    } else if (which == "a") {
        n_a = n;
        cuda_err = hipMemcpyToSymbol(HIP_SYMBOL(d_n_a), &n_a, sizeof(double));
        if (cuda_err != hipSuccess) {
            std::cerr << "Particle::setExponent: Error copying n_a to device: " << hipGetErrorString(cuda_err) << std::endl;
            exit(EXIT_FAILURE);  // TODO: make this a function and put it in a cuda module
        }
    } else if (which == "b") {
        n_b = n;
        cuda_err = hipMemcpyToSymbol(HIP_SYMBOL(d_n_b), &n_b, sizeof(double));
        if (cuda_err != hipSuccess) {
            std::cerr << "Particle::setExponent: Error copying n_b to device: " << hipGetErrorString(cuda_err) << std::endl;
            exit(EXIT_FAILURE);  // TODO: make this a function and put it in a cuda module
        }
    } else if (which == "l") {
        n_l = n;
        cuda_err = hipMemcpyToSymbol(HIP_SYMBOL(d_n_l), &n_l, sizeof(double));
        if (cuda_err != hipSuccess) {
            std::cerr << "Particle::setExponent: Error copying n_l to device: " << hipGetErrorString(cuda_err) << std::endl;
            exit(EXIT_FAILURE);  // TODO: make this a function and put it in a cuda module
        }
    } else {
        throw std::invalid_argument("Particle::setExponent: which must be 'c', 'a', 'b', or 'l', not " + which);
    }
}

void Particle::setAllExponents(double n_c, double n_a, double n_b, double n_l) {
    setExponent(n_c, "c");
    setExponent(n_a, "a");
    setExponent(n_b, "b");
    setExponent(n_l, "l");
}

double Particle::getExponent(std::string which) {
    if (which == "c") {
        return n_c;
    } else if (which == "a") {
        return n_a;
    } else if (which == "b") {
        return n_b;
    } else if (which == "l") {
        return n_l;
    } else {
        throw std::invalid_argument("Particle::getExponent: which must be 'c', 'a', 'b', or 'l', not " + which);
    }
}

void Particle::initializeBox(double packing_fraction) {
    // set the box size to an arbitrary initial value
    double side_length = 1.0;
    thrust::host_vector<double> host_box_size(N_DIM, side_length);
    setBoxSize(host_box_size);
    // then rescale the box size to the desired packing fraction
    scaleToPackingFraction(packing_fraction);
}

void Particle::setRandomUniform(thrust::device_vector<double>& values, double min, double max, long seed_offset) {
    thrust::counting_iterator<long> index_sequence_begin(seed + seed_offset);
    thrust::transform(index_sequence_begin, index_sequence_begin + values.size(), values.begin(), RandomUniform(min, max, seed));
}

void Particle::setRandomNormal(thrust::device_vector<double>& values, double mean, double stddev, long seed_offset) {
    thrust::counting_iterator<long> index_sequence_begin(seed + seed_offset);
    thrust::transform(index_sequence_begin, index_sequence_begin + values.size(), values.begin(), RandomNormal(mean, stddev, seed));
}

void Particle::setRandomPositions() {
    thrust::host_vector<double> host_box_size = box_size.getData();
    positions.fillRandomUniform(0.0, host_box_size[0], 0.0, host_box_size[1], 1, seed);
}

void Particle::removeMeanVelocities() {
    double mean_vel_x = thrust::reduce(velocities.x.d_vec.begin(), velocities.x.d_vec.end()) / velocities.x.d_vec.size();
    double mean_vel_y = thrust::reduce(velocities.y.d_vec.begin(), velocities.y.d_vec.end()) / velocities.y.d_vec.size();
    kernelRemoveMeanVelocities<<<particle_dim_grid, particle_dim_block>>>(velocities.x.d_ptr, velocities.y.d_ptr, mean_vel_x, mean_vel_y);
}

void Particle::scaleVelocitiesToTemperature(double temperature) {
    double current_temp = calculateTemperature();
    if (current_temp <= 0.0) {
        std::cout << "WARNING: Particle::scaleVelocitiesToTemperature: Current temperature is " << current_temp << ", there will be an error!" << std::endl;
    }
    double scale_factor = std::sqrt(temperature / current_temp);
    velocities.scale(scale_factor, scale_factor);
}

void Particle::setRandomVelocities(double temperature) {
    velocities.fillRandomNormal(0.0, std::sqrt(temperature), 0.0, std::sqrt(temperature), 1, seed);
    removeMeanVelocities();
    scaleVelocitiesToTemperature(temperature);
}

double Particle::getDiameter(std::string which) {
    if (which == "min") {
        return 2.0 * *thrust::min_element(radii.d_vec.begin(), radii.d_vec.end());
    } else if (which == "max") {
        return 2.0 * *thrust::max_element(radii.d_vec.begin(), radii.d_vec.end());
    } else if (which == "mean") {
        return 2.0 * thrust::reduce(radii.d_vec.begin(), radii.d_vec.end()) / radii.d_vec.size();
    } else {
        throw std::invalid_argument("Particle::getDiameter: which must be 'min', 'max', or 'mean', not " + which);
    }
}

void Particle::setBiDispersity(double size_ratio, double count_ratio) {
    if (size_ratio < 1.0) {
        throw std::invalid_argument("Particle::setBiDispersity: size_ratio must be > 1.0");
    }
    if (count_ratio < 0.0 || count_ratio > 1.0) {
        throw std::invalid_argument("Particle::setBiDispersity: count_ratio must be < 1.0 and > 0.0");
    }
    thrust::host_vector<double> host_radii(n_particles);
    long n_large = static_cast<long>(n_particles * count_ratio);
    double diam_large = size_ratio;
    double diam_small = 1.0;
    for (long i = 0; i < n_large; i++) {
        host_radii[i] = diam_large / 2.0;
    }
    for (long i = n_large; i < n_particles; i++) {
        host_radii[i] = diam_small / 2.0;
    }
    radii.setData(host_radii);
}

double Particle::getBoxArea() {
    return thrust::reduce(box_size.d_vec.begin(), box_size.d_vec.end(), 1.0, thrust::multiplies<double>());
}

double Particle::getPackingFraction() {
    double box_area = getBoxArea();
    double area = getArea();
    return area / box_area;
}

double Particle::getDensity() {
    return getPackingFraction() - getOverlapFraction();
}

void Particle::scaleToPackingFraction(double packing_fraction) {
    double new_side_length = std::pow(getArea() / packing_fraction, 1.0 / N_DIM);
    double side_length = std::pow(getBoxArea(), 1.0 / N_DIM);
    double scale_factor = new_side_length / side_length;
    positions.scale(scale_factor, scale_factor);
    thrust::host_vector<double> host_box_size(N_DIM, new_side_length);
    setBoxSize(host_box_size);
}

double Particle::totalKineticEnergy() const {
    return thrust::reduce(kinetic_energy.d_vec.begin(), kinetic_energy.d_vec.end(), 0.0, thrust::plus<double>());
}

double Particle::totalPotentialEnergy() const {
    return thrust::reduce(potential_energy.d_vec.begin(), potential_energy.d_vec.end(), 0.0, thrust::plus<double>());
}

double Particle::totalEnergy() const {
    return totalKineticEnergy() + totalPotentialEnergy();
}

void Particle::scalePositions(double scale_factor) {
    positions.scale(scale_factor, scale_factor);
}

void Particle::updatePositions(double dt) {
    kernelUpdatePositions<<<particle_dim_grid, particle_dim_block>>>(positions.x.d_ptr, positions.y.d_ptr, last_neigh_positions.x.d_ptr, last_neigh_positions.y.d_ptr, last_cell_positions.x.d_ptr, last_cell_positions.y.d_ptr, neigh_displacements_sq.d_ptr, cell_displacements_sq.d_ptr, velocities.x.d_ptr, velocities.y.d_ptr, dt);
}

void Particle::updateVelocities(double dt) {
    kernelUpdateVelocities<<<particle_dim_grid, particle_dim_block>>>(velocities.x.d_ptr, velocities.y.d_ptr, forces.x.d_ptr, forces.y.d_ptr, masses.d_ptr, dt);

}

double Particle::getMaxSquaredNeighborDisplacement() {
    return thrust::reduce(neigh_displacements_sq.d_vec.begin(), neigh_displacements_sq.d_vec.end(), 0.0, thrust::maximum<double>());
}

double Particle::getMaxSquaredCellDisplacement() {
    return thrust::reduce(cell_displacements_sq.d_vec.begin(), cell_displacements_sq.d_vec.end(), 0.0, thrust::maximum<double>());
}

void Particle::updateVerletList() {
    neighbor_list.fill(-1L);
    kernelUpdateNeighborList<<<particle_dim_grid, particle_dim_block>>>(positions.x.d_ptr, positions.y.d_ptr, last_neigh_positions.x.d_ptr, last_neigh_positions.y.d_ptr, neigh_displacements_sq.d_ptr, neighbor_cutoff);
    max_neighbors = thrust::reduce(num_neighbors.d_vec.begin(), num_neighbors.d_vec.end(), -1L, thrust::maximum<long>());
    if (max_neighbors > max_neighbors_allocated) {
        max_neighbors_allocated = std::pow(2, std::ceil(std::log2(max_neighbors)));
        std::cout << "Particle::updateVerletList: Resizing neighbor list to " << max_neighbors_allocated << std::endl;
        neighbor_list.resize(n_particles * max_neighbors_allocated);
        neighbor_list.fill(-1L);
        syncNeighborList();
        kernelUpdateNeighborList<<<particle_dim_grid, particle_dim_block>>>(positions.x.d_ptr, positions.y.d_ptr, last_neigh_positions.x.d_ptr, last_neigh_positions.y.d_ptr, neigh_displacements_sq.d_ptr, neighbor_cutoff);
    }
}

void Particle::checkForAllToAllUpdate() {
    // Do nothing
}

void Particle::checkForNeighborUpdate() {
    (this->*checkForNeighborUpdatePtr)();
}

void Particle::checkForVerletListUpdate() {
    double tolerance = 3.0;
    double max_squared_neighbor_displacement = getMaxSquaredNeighborDisplacement();
    if (tolerance * max_squared_neighbor_displacement > neighbor_displacement_threshold_sq) {
        updateVerletList();
    }
}

void Particle::checkForCellListUpdate() {
    double tolerance = 3.0;
    double max_squared_cell_displacement = getMaxSquaredCellDisplacement();
    if (tolerance * max_squared_cell_displacement > cell_displacement_threshold_sq) {
        updateCellList();
        updateCellNeighborList();
    } else {
        double max_squared_neighbor_displacement = getMaxSquaredNeighborDisplacement();
        if (tolerance * max_squared_neighbor_displacement > neighbor_displacement_threshold_sq) {
            updateCellNeighborList();
        }
    }
}

void Particle::initNeighborList() {
    (this->*initNeighborListPtr)();
}

void Particle::initVerletListVariables() {
    neighbor_list.resizeAndFill(n_particles * max_neighbors_allocated, -1L);
    num_neighbors.resizeAndFill(n_particles, 0L);
    last_neigh_positions.resizeAndFill(n_particles, 0.0, 0.0);
    neigh_displacements_sq.resizeAndFill(n_particles, 0.0);
    last_cell_positions.resizeAndFill(n_particles, 0.0, 0.0);  // TODO: this is a waste of memory for non-cell list usage but would require defining a new position update kernel
    cell_displacements_sq.resizeAndFill(n_particles, 0.0);
}

void Particle::initVerletList() {
    initVerletListVariables();
    syncNeighborList();
    updateVerletList();
}

void Particle::initAllToAllListVariables() {
    this->max_neighbors_allocated = n_particles;
    initVerletListVariables();
}

void Particle::initAllToAllList() {
    initAllToAllListVariables();
    syncNeighborList();
    updateVerletList();
}

void Particle::initCellListVariables() {
    cell_index.resizeAndFill(n_particles, -1L);
    particle_index.resize(n_particles);
    static_particle_index.resize(n_particles);
    cell_start.resize(n_cells + 1);
    thrust::sequence(particle_index.d_vec.begin(), particle_index.d_vec.end());
    thrust::sequence(static_particle_index.d_vec.begin(), static_particle_index.d_vec.end());
}

void Particle::initCellList() {
    initVerletListVariables();
    syncNeighborList();
    initCellListVariables();
    updateCellList();
    updateCellNeighborList();
}

bool Particle::setNeighborSize(double neighbor_cutoff_multiplier, double neighbor_displacement_multiplier) {
    this->max_neighbors_allocated = 4;  // initial assumption, probably could be refined
    this->neighbor_cutoff = neighbor_cutoff_multiplier * getDiameter("max");
    this->neighbor_displacement_threshold_sq = std::pow(neighbor_displacement_multiplier * neighbor_cutoff, 2);
    thrust::host_vector<double> host_box_size = box_size.getData();
    double box_diagonal = std::sqrt(host_box_size[0] * host_box_size[0] + host_box_size[1] * host_box_size[1]);
    if (neighbor_cutoff >= box_diagonal) {
        std::cout << "Particle::setNeighborSize: Neighbor radius exceeds the box size" << std::endl;
        return false;
    }
    return true;
}

bool Particle::setCellSize(double num_particles_per_cell, double cell_displacement_multiplier) {
    long min_num_cells_dim = 4;  // if there are fewer than 4 cells in one axis, the cell list is spiritually defeated
    double number_density = getNumberDensity();
    double trial_cell_size = std::sqrt(num_particles_per_cell / number_density);
    double min_cell_size = 2.0 * getDiameter("max");  // somewhat arbitrary bound, probably could be refined
    thrust::host_vector<double> host_box_size = box_size.getData();
    n_cells_dim = static_cast<long>(std::floor(host_box_size[0] / trial_cell_size));
    n_cells = n_cells_dim * n_cells_dim;
    if (n_cells_dim < min_num_cells_dim) {
        std::cout << "Particle::setCellSize: fewer than " << min_num_cells_dim << " cells in one dimension" << std::endl;
        n_cells_dim = min_num_cells_dim;
        n_cells = n_cells_dim * n_cells_dim;
    }
    cell_size = host_box_size[0] / n_cells_dim;
    if (cell_size < min_cell_size) {
        std::cout << "Particle::setCellSize: cell size is less than twice the maximum diameter" << std::endl;  // 
        cell_size = min_cell_size;

        // try to make the cell again
        n_cells_dim = static_cast<long>(std::floor(host_box_size[0] / cell_size));
        n_cells = n_cells_dim * n_cells_dim;
        if (n_cells_dim < min_num_cells_dim) {
            std::cout << "Particle::setCellSize: Failed to make cell list - fewer than " << min_num_cells_dim << " cells in one dimension and cell size is less than twice the maximum particle diameter" << std::endl;
            return false;
        }
    }
    cell_displacement_threshold_sq = std::pow(cell_displacement_multiplier * cell_size, 2);
    std::cout << "Particle::setCellSize: Cell size set to " << cell_size << " and cell displacement set to " << cell_displacement_threshold_sq << " for " << n_cells << " cells" << std::endl;
    syncCellList();
    return true;
}

void Particle::syncCellList() {
    hipError_t cuda_err = hipMemcpyToSymbol(HIP_SYMBOL(d_n_cells), &n_cells, sizeof(long));
    if (cuda_err != hipSuccess) {
        std::cerr << "Particle::syncCellList: Error copying n_cells to device: " << hipGetErrorString(cuda_err) << std::endl;
        exit(EXIT_FAILURE);
    }
    cuda_err = hipMemcpyToSymbol(HIP_SYMBOL(d_n_cells_dim), &n_cells_dim, sizeof(long));
    if (cuda_err != hipSuccess) {
        std::cerr << "Particle::syncCellList: Error copying n_cells_dim to device: " << hipGetErrorString(cuda_err) << std::endl;
        exit(EXIT_FAILURE);
    }
    cuda_err = hipMemcpyToSymbol(HIP_SYMBOL(d_cell_size), &cell_size, sizeof(double));
    if (cuda_err != hipSuccess) {
        std::cerr << "Particle::syncCellList: Error copying cell_size to device: " << hipGetErrorString(cuda_err) << std::endl;
        exit(EXIT_FAILURE);
    }
}

void Particle::reorderParticleData() {
    thrust::sort_by_key(cell_index.d_vec.begin(), cell_index.d_vec.end(), thrust::make_zip_iterator(thrust::make_tuple(particle_index.d_vec.begin(), static_particle_index.d_vec.begin())));
    kernelReorderParticleData<<<particle_dim_grid, particle_dim_block>>>(particle_index.d_ptr, positions.x.d_ptr, positions.y.d_ptr, forces.x.d_ptr, forces.y.d_ptr, velocities.x.d_ptr, velocities.y.d_ptr, masses.d_ptr, radii.d_ptr, positions.x.d_temp_ptr, positions.y.d_temp_ptr, forces.x.d_temp_ptr, forces.y.d_temp_ptr, velocities.x.d_temp_ptr, velocities.y.d_temp_ptr, masses.d_temp_ptr, radii.d_temp_ptr, last_cell_positions.x.d_ptr, last_cell_positions.y.d_ptr, cell_displacements_sq.d_ptr);
    positions.swap();
    forces.swap();
    velocities.swap();
    masses.swap();
    radii.swap();
}

void Particle::updateCellList() {
    cell_start.d_vec[n_cells] = n_particles;
    kernelGetCellIndexForParticle<<<particle_dim_grid, particle_dim_block>>>(positions.x.d_ptr, positions.y.d_ptr, cell_index.d_ptr, particle_index.d_ptr);
    reorderParticleData();
    hipDeviceSynchronize();
    // TODO: this is a kernel over cells - could probably be parallelized better
    long width_offset = 2;
    long width = n_particles / n_cells;
    // TODO FIXXXXXX
    kernelGetFirstParticleIndexForCell<<<n_cells, particle_dim_block>>>(cell_index.d_ptr, cell_start.d_ptr, width_offset, width);
}

// TODO: look into better ways to structure the grid and block sizes
void Particle::updateCellNeighborList() {
    neighbor_list.fill(-1L);
    kernelUpdateCellNeighborList<<<particle_dim_grid, particle_dim_block>>>(positions.x.d_ptr, positions.y.d_ptr, last_neigh_positions.x.d_ptr, last_neigh_positions.y.d_ptr, neighbor_cutoff, cell_index.d_ptr, cell_start.d_ptr, neigh_displacements_sq.d_ptr);
    max_neighbors = thrust::reduce(num_neighbors.d_vec.begin(), num_neighbors.d_vec.end(), -1L, thrust::maximum<long>());
    if (max_neighbors > max_neighbors_allocated) {
        max_neighbors_allocated = std::pow(2, std::ceil(std::log2(max_neighbors)));
        std::cout << "Particle::updateCellNeighborList: Resizing neighbor list to " << max_neighbors_allocated << std::endl;
        neighbor_list.resizeAndFill(n_particles * max_neighbors_allocated, -1L);
        syncNeighborList();
        kernelUpdateCellNeighborList<<<particle_dim_grid, particle_dim_block>>>(positions.x.d_ptr, positions.y.d_ptr, last_neigh_positions.x.d_ptr, last_neigh_positions.y.d_ptr, neighbor_cutoff, cell_index.d_ptr, cell_start.d_ptr, neigh_displacements_sq.d_ptr);
    }
}

// TODO: this should be a single kernel
void Particle::zeroForceAndPotentialEnergy() {
    kernelZeroForceAndPotentialEnergy<<<particle_dim_grid, particle_dim_block>>>(forces.x.d_ptr, forces.y.d_ptr, potential_energy.d_ptr);
}

double Particle::calculateTemperature() {
    calculateKineticEnergy();
    return totalKineticEnergy() * 2.0 / n_dof;
}

double Particle::getTimeUnit() {
    double average_mass = thrust::reduce(masses.d_vec.begin(), masses.d_vec.end()) / n_particles;
    return getDiameter("min") * std::sqrt(average_mass / getEnergyScale("c"));
}

void Particle::setMass(double mass) {
    masses.fill(mass);
}


double Particle::getNumberDensity() {
    return n_particles / getBoxArea();
}
