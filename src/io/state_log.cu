#include "hip/hip_runtime.h"
#include "../../include/io/state_log.h"
#include "../../include/io/io_utils.h"
#include <thrust/host_vector.h>

StateLog::StateLog(LogGroupConfigDict config, Orchestrator& orchestrator, const std::string& root, const std::string& indexed_file_prefix, const std::string& extension)
    : BaseLogGroup(config, orchestrator), root(root), indexed_file_prefix(indexed_file_prefix), extension(extension) {
    this->parallel = true;
}

StateLog::~StateLog() {
}

// need a key: value pair for each reorder array

void StateLog::gather_data(long step) {
    if (orchestrator.arrays_need_reordering) {
        reorder_index_data = orchestrator.get_reorder_index_data();
    }
    for (const auto& name : log_names) {
        ArrayData array_data = orchestrator.get_array_data(name);
        gathered_data[name] = array_data;
    }
}

void StateLog::write_values(const std::filesystem::path& root) {
    for (auto& [name, array_data] : gathered_data) {
        std::filesystem::path file_path = root / (name + extension);
        if (orchestrator.arrays_need_reordering && reorder_index_data.find(array_data.index_array_name) != reorder_index_data.end()) {
            reorder_array(array_data, reorder_index_data[array_data.index_array_name]);
        }
        write_array_data_to_file(file_path.string(), array_data, precision);
    }
}


void StateLog::log(long step) {
    std::filesystem::path timestep_root_path = std::filesystem::path(root) / (indexed_file_prefix + std::to_string(step));
    make_dir(timestep_root_path.string(), true);
    write_values(timestep_root_path);
}

void StateLog::write_state_to_path(const std::filesystem::path& path) {
    write_values(path);
}