#include "../../include/constants.h"
#include "../../include/functors.h"
#include "../../include/particle/particle.h"
#include "../../include/particle/disk.h"
#include "../../include/kernels/kernels.cuh"
#include <iostream>
#include <string>
#include <vector>
#include <cmath>
#include <random>
#include <time.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/fill.h>
#include <thrust/copy.h>
#include <thrust/transform.h>
#include <thrust/transform_reduce.h>
#include <thrust/inner_product.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/permutation_iterator.h>
#include <thrust/functional.h>

Disk::Disk() {
    std::cout << "Disk::Disk: Start" << std::endl;
    std::cout << "Disk::Disk: End" << std::endl;
}

Disk::~Disk() {
    std::cout << "Disk::~Disk: Start" << std::endl;
    std::cout << "Disk::~Disk: End" << std::endl;
}

// ----------------------------------------------------------------------
// --------------------- Overridden Methods -----------------------------
// ----------------------------------------------------------------------


void Disk::setKernelDimensions(long dim_block) {
    int maxThreadsPerBlock;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    maxThreadsPerBlock = deviceProp.maxThreadsPerBlock;
    std::cout << "CUDA Info: Particle::setKernelDimensions: Max threads per block: " << maxThreadsPerBlock << std::endl;
    if (dim_block > maxThreadsPerBlock) {
        std::cout << "WARNING: Particle::setKernelDimensions: dim_block exceeds maxThreadsPerBlock, adjusting to maxThreadsPerBlock" << std::endl;
        dim_block = maxThreadsPerBlock;
    }
    if (n_particles <= 0) {
        std::cout << "ERROR: Disk::setKernelDimensions: n_particles is 0.  Set n_particles before setting kernel dimensions." << std::endl;
        exit(EXIT_FAILURE);
    }

    if (n_particles < dim_block) {
        dim_block = n_particles;
    }
    this->dim_block = dim_block;
    this->dim_grid = (n_particles + dim_block - 1) / dim_block;

    if (n_vertices > 0) {
        std::cout << "WARNING: Disk::setKernelDimensions: n_vertices is " << n_vertices << ".  This is being ignored." << std::endl;
    }

    syncKernelDimensions();
}

// ----------------------------------------------------------------------
// ------------- Implementation of Pure Virtual Methods -----------------
// ----------------------------------------------------------------------

double Disk::getArea() const {
    return thrust::transform_reduce(d_radii.begin(), d_radii.end(), Square(), 0.0, thrust::plus<double>()) * PI;
}

double Disk::getOverlapFraction() const {
    // std::cout << "FIXME: Implement getOverlapFraction" << std::endl;
    // std::cout << "FIXME: Implement getOverlapFraction" << std::endl;
    // std::cout << "FIXME: Implement getOverlapFraction" << std::endl;
    // std::cout << "FIXME: Implement getOverlapFraction" << std::endl;
    // std::cout << "FIXME: Implement getOverlapFraction" << std::endl;
    return 0.0;
}

void Disk::calculateForces() {
    kernelCalcDiskForces<<<dim_grid, dim_block>>>(d_positions_ptr, d_radii_ptr, d_forces_ptr, d_potential_energy_ptr);
}

void Disk::calculateKineticEnergy() {
    kernelCalculateTranslationalKineticEnergy<<<dim_grid, dim_block>>>(d_velocities_ptr, d_masses_ptr, d_kinetic_energy_ptr);
}