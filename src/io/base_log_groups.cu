#include "hip/hip_runtime.h"
#include "../../include/io/base_log_groups.h"
#include "../../include/io/orchestrator.h"
#include <string>
#include <iostream>


LogGroupConfig config_from_names_lin(std::vector<std::string> log_names, long num_steps, long num_saves, std::string group_name) {
    LogGroupConfig config;
    config.log_names = log_names;
    config.save_style = "lin";
    config.save_freq = static_cast<long>(num_steps / num_saves);
    config.group_name = group_name;
    return config;
}

LogGroupConfig config_from_names_log(std::vector<std::string> log_names, long num_steps, long num_saves, long min_save_decade, std::string group_name) {
    LogGroupConfig config;
    config.log_names = log_names;
    config.save_style = "log";
    config.reset_save_decade = static_cast<long>(num_steps / num_saves);
    config.min_save_decade = min_save_decade;
    config.group_name = group_name;
    return config;
}

LogGroupConfig config_from_names_lin_everyN(std::vector<std::string> log_names, long save_freq, std::string group_name) {
    LogGroupConfig config;
    config.log_names = log_names;
    config.save_style = "lin";
    config.save_freq = save_freq;
    config.group_name = group_name;
    return config;
}

BaseLogGroup::BaseLogGroup(LogGroupConfig config, Orchestrator& orchestrator) : config(config), orchestrator(orchestrator) {
}

BaseLogGroup::~BaseLogGroup() {
}

void BaseLogGroup::update_log_status(long step) {
    if (config.save_style == "lin") {
        should_log = step % config.save_freq == 0;
    } else if (config.save_style == "log") {
        if (step > config.multiple * config.reset_save_decade) {
            config.save_freq = config.min_save_decade;
            config.multiple += 1;
        }
        if ((step - (config.multiple - 1) * config.reset_save_decade) > config.save_freq * config.decade) {
            config.save_freq *= config.decade;
        }
        if ((step - (config.multiple - 1) * config.reset_save_decade) % config.save_freq == 0) {
            should_log = true;
        } else {
            should_log = false;
        }
    } else {
        std::cout << "ERROR: LogManager::update_log_status: Invalid save style: " << config.save_style << std::endl;
        exit(1);
    }
}





MacroLog::MacroLog(LogGroupConfig config, Orchestrator& orchestrator) : BaseLogGroup(config, orchestrator) {
    unmodified_log_names = get_unmodified_log_names();
}

MacroLog::~MacroLog() {
}

bool MacroLog::log_name_is_modified(std::string log_name) {
    return log_name.find(modifier) != std::string::npos;
}

std::vector<std::string> MacroLog::get_unmodified_log_names() {
    std::vector<std::string> unmodified_log_names;
    for (auto& log_name : config.log_names) {
        size_t pos = log_name.find(modifier);
        if (pos != std::string::npos) {
            unmodified_log_names.push_back(log_name.substr(0, pos));
        } else {
            unmodified_log_names.push_back(log_name);
        }
    }
    return unmodified_log_names;
}

std::string MacroLog::get_modifier(std::string log_name) {
    size_t pos = log_name.find(modifier);
    if (pos != std::string::npos) {
        return log_name.substr(pos + 1);
    } else {
        return "";
    }
}