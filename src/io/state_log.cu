#include "hip/hip_runtime.h"
#include "../../include/io/state_log.h"
#include "../../include/io/utils.h"
#include <thrust/host_vector.h>

StateLog::StateLog(LogGroupConfig config, Orchestrator& orchestrator, const std::string& root_path, const std::string& indexed_file_prefix, const std::string& extension)
    : BaseLogGroup(config, orchestrator), root_path(root_path), indexed_file_prefix(indexed_file_prefix), extension(extension) {
}

StateLog::~StateLog() {
}

void StateLog::write_values(std::filesystem::path root_path) {
    for (int i = 0; i < config.log_names.size(); i++) {
        thrust::host_vector<double> value = orchestrator.get_vector_value<double>(config.log_names[i]);
        std::vector<long> size = orchestrator.get_vector_size(config.log_names[i]);
        std::filesystem::path file_path = root_path / (config.log_names[i] + extension);
        write_array_to_file(
            file_path.string(),
            value,
            size[0],
            size[1],
            precision
        );
    }
}

void StateLog::log(long step) {
    std::filesystem::path timestep_root_path = std::filesystem::path(root_path) / (indexed_file_prefix + std::to_string(step));
    make_dir(timestep_root_path.string(), true);
    write_values(timestep_root_path);
}

void StateLog::write_state() {
    write_values(root_path);
}