#include "hip/hip_runtime.h"
#include "../../include/io/io_manager.h"
#include <iostream>
#include <filesystem>
#include <thread>
#include <vector>

IOManager::IOManager(std::vector<LogGroupConfigDict> log_configs, Particle& particle, Integrator* integrator, std::string root, long num_threads, bool overwrite) : particle(particle), integrator(integrator), orchestrator(particle, integrator), root(root), num_threads(num_threads), overwrite(overwrite), log_configs(log_configs), thread_pool(num_threads) {
    root_path = std::filesystem::path(root);
    if (!root_path.empty()) {
        if (overwrite) {
            std::filesystem::remove_all(root_path);
            std::filesystem::create_directories(root_path);
        } else {
            std::cerr << "ERROR: IOManager::IOManager: root path " << root_path << " already exists and overwriting is disabled!" << std::endl;
            return;
        }
    }
    
    use_parallel = num_threads > 1;

    for (auto& config : log_configs) {

        if (config["group_name"] == "energy") {
            if (system_dir_path.empty()) {
                init_path(system_dir_path, system_dir_name);
                make_dir(system_dir_path, overwrite);  // may need to change function signature
            }
            std::filesystem::path energy_file_path = system_dir_path / (energy_file_name + energy_file_extension);
            log_groups.push_back(new EnergyLog(config, orchestrator, energy_file_path, overwrite));

        } else if (config["group_name"] == "console") {
            log_groups.push_back(new ConsoleLog(config, orchestrator));

        } else if (config["group_name"] == "state") {
            if (trajectory_dir_path.empty()) {
                init_path(trajectory_dir_path, trajectory_dir_name);
                make_dir(trajectory_dir_path, overwrite);  // may need to change function signature
            }
            log_groups.push_back(new StateLog(config, orchestrator, trajectory_dir_path, indexed_file_prefix, state_file_extension));
        
        } else if (config["group_name"] == "init") {
            if (system_dir_path.empty()) {
                init_path(system_dir_path, system_dir_name);
                make_dir(system_dir_path, overwrite);  // may need to change function signature
            }
            state_log = new StateLog(config, orchestrator, system_dir_path, "", state_file_extension);

        } else {
            std::cerr << "ERROR: IOManager::IOManager:" << config["group_name"] << " is not a valid log group name" << std::endl;
        }
    }

    // define the dependencies in the log groups
    for (auto& log_group : log_groups) {
        log_group->define_dependencies();
    }

    if (state_log != nullptr) {
        // make the init and restart directories
        init_dir_path = system_dir_path / init_dir_name;
        make_dir(init_dir_path.string(), true);
        restart_dir_path = system_dir_path / restart_dir_name;
        make_dir(restart_dir_path.string(), true);

        // gather the data
        state_log->gather_data(0);
        // write the data
        state_log->write_state_to_path(init_dir_path);
    }
}

IOManager::~IOManager() {
    thread_pool.shutdown();
    for (auto& log_group : log_groups) {
        delete log_group;
    }
}

void IOManager::write_restart_file(long step) {
    // write the state to the restart directory
    state_log->write_state_to_path(restart_dir_path);
    // write the current step to a file using the write_json_to_file function
    write_json_to_file(restart_dir_path / "current_step.json", nlohmann::json{{"step", step}});
}

void IOManager::init_path(std::filesystem::path& path, const std::string& path_name) {
    if (root.empty()) {
        std::cerr << "ERROR: IOManager::init_path:" << path_name << " root is empty" << std::endl;
        return;
    }
    path = static_cast<std::filesystem::path>(root) / static_cast<std::filesystem::path>(path_name);
}

void IOManager::log(long step, bool force) {
    // figure out if any logs need to be written
    bool log_required = false;
    for (BaseLogGroup* log_group : log_groups) {
        log_group->update_log_status(step);
        if (log_group->should_log || force) {
            log_required = true;
        }
    }

    // do the logging

    if (log_required) {
        // handle dependency calculation if any
        orchestrator.reset_dependency_status();
        for (BaseLogGroup* log_group : log_groups) {
            if (log_group->has_dependencies) {
                log_group->handle_dependencies();
            }
        }

        // gather the data
        for (BaseLogGroup* log_group : log_groups) {
            if (log_group->should_log || force) {
                log_group->gather_data(step);
            }
        }

        // now we can disconnect from the simulation and run these in parallel

        // log
        std::vector<std::thread> threads;  // Store threads for async log groups
        for (BaseLogGroup* log_group : log_groups) {
            if (log_group->should_log || force) {
                if (log_group->parallel && use_parallel) {
                    thread_pool.enqueue([log_group, step]() {
                        log_group->log(step);
                    });
                } else {
                    log_group->log(step);
                }
            }
        }

        // Detach all threads
        for (auto& thread : threads) {
            thread.detach();  // Let them run independently
        }
    }
}

void IOManager::write_log_configs(std::filesystem::path path) {
    for (auto& config : log_configs) {
        std::string group_name = config["group_name"].get<std::string>();
        config.to_json(path / (group_name + "_log_config.json"));
    }
}

void IOManager::write_particle_config(std::filesystem::path path) {
    particle.config.to_json(path / "particle_config.json");
}

void IOManager::write_integrator_config(std::filesystem::path path) {
    integrator->config.to_json(path / "integrator_config.json");
}

void IOManager::write_params() {
    if (system_dir_path.empty()) {
        init_path(system_dir_path, system_dir_name);
        make_dir(system_dir_path, overwrite);  // may need to change function signature
    }
    write_log_configs(system_dir_path);
    write_particle_config(system_dir_path);
    if (integrator != nullptr) {
        write_integrator_config(system_dir_path);
    }
    // TODO: write run params
}

// void IOManager::write_state_to_path() {
//     if (state_log == nullptr) {
//         std::cerr << "ERROR: IOManager::write_state_to_path: state_log is not initialized" << std::endl;
//         return;
//     }
//     state_log->write_state_to_path();
// }