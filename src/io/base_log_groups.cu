#include "hip/hip_runtime.h"
#include "../../include/io/base_log_groups.h"
#include "../../include/io/orchestrator.h"
#include <string>
#include <iostream>

BaseLogGroup::BaseLogGroup(LogGroupConfig config, Orchestrator& orchestrator) : config(config), orchestrator(orchestrator) {
}

BaseLogGroup::~BaseLogGroup() {
}

void BaseLogGroup::update_log_status(long step) {
    if (config.save_style == "lin") {
        should_log = step % config.save_freq == 0;
    } else if (config.save_style == "log") {
        if (step > config.multiple * config.reset_save_decade) {
            config.save_freq = config.min_save_decade;
            config.multiple += 1;
        }
        if ((step - (config.multiple - 1) * config.reset_save_decade) > config.save_freq * config.decade) {
            config.save_freq *= config.decade;
        }
        if ((step - (config.multiple - 1) * config.reset_save_decade) % config.save_freq == 0) {
            should_log = true;
        } else {
            should_log = false;
        }
    } else {
        std::cout << "ERROR: LogManager::update_log_status: Invalid save style: " << config.save_style << std::endl;
        exit(1);
    }
}





MacroLog::MacroLog(LogGroupConfig config, Orchestrator& orchestrator) : BaseLogGroup(config, orchestrator) {
    unmodified_log_names = get_unmodified_log_names();
}

MacroLog::~MacroLog() {
}

bool MacroLog::log_name_is_modified(std::string log_name) {
    return log_name.find(modifier) != std::string::npos;
}

std::vector<std::string> MacroLog::get_unmodified_log_names() {
    std::vector<std::string> unmodified_log_names;
    for (auto& log_name : config.log_names) {
        size_t pos = log_name.find(modifier);
        if (pos != std::string::npos) {
            unmodified_log_names.push_back(log_name.substr(0, pos));
        } else {
            unmodified_log_names.push_back(log_name);
        }
    }
    return unmodified_log_names;
}

std::string MacroLog::get_modifier(std::string log_name) {
    size_t pos = log_name.find(modifier);
    if (pos != std::string::npos) {
        return log_name.substr(pos + 1);
    } else {
        return "";
    }
}