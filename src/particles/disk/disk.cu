#include "../../include/constants.h"
#include "../../include/functors.h"
#include "../../include/particles/base/particle.h"
#include "../../include/io/io_utils.h"
#include "../../include/particles/disk/disk.h"
#include "../../include/particles/disk/kernels.cuh"
#include <iostream>
#include <string>
#include <vector>
#include <cmath>
#include <random>
#include <time.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/fill.h>
#include <thrust/copy.h>
#include <thrust/transform.h>
#include <thrust/transform_reduce.h>
#include <thrust/inner_product.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/permutation_iterator.h>
#include <thrust/functional.h>

Disk::Disk() {
}

Disk::~Disk() {
}

// ----------------------------------------------------------------------
// --------------------- Overridden Methods -----------------------------
// ----------------------------------------------------------------------


void Disk::setKernelDimensions(long particle_dim_block) {
    int maxThreadsPerBlock;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    maxThreadsPerBlock = deviceProp.maxThreadsPerBlock;
    std::cout << "CUDA Info: Particle::setKernelDimensions: Max threads per block: " << maxThreadsPerBlock << std::endl;
    if (particle_dim_block > maxThreadsPerBlock) {
        std::cout << "WARNING: Particle::setKernelDimensions: particle_dim_block exceeds maxThreadsPerBlock, adjusting to maxThreadsPerBlock" << std::endl;
        particle_dim_block = maxThreadsPerBlock;
    }
    if (n_particles <= 0) {
        std::cout << "ERROR: Disk::setKernelDimensions: n_particles is 0.  Set n_particles before setting kernel dimensions." << std::endl;
        exit(EXIT_FAILURE);
    }

    if (n_particles < particle_dim_block) {
        particle_dim_block = n_particles;
    }
    this->particle_dim_block = particle_dim_block;
    this->particle_dim_grid = (n_particles + particle_dim_block - 1) / particle_dim_block;

    if (n_vertices > 0) {
        std::cout << "WARNING: Disk::setKernelDimensions: n_vertices is " << n_vertices << ".  This is being ignored." << std::endl;
    }

    syncKernelDimensions();
}

// ----------------------------------------------------------------------
// ------------- Implementation of Pure Virtual Methods -----------------
// ----------------------------------------------------------------------


double Disk::getParticleArea() const {
    return thrust::transform_reduce(radii.d_vec.begin(), radii.d_vec.end(), Square(), 0.0, thrust::plus<double>()) * PI;
}

double Disk::getOverlapFraction() const {
    std::cout << "FIXME: Implement getOverlapFraction" << std::endl;
    // std::cout << "FIXME: Implement getOverlapFraction" << std::endl;
    // std::cout << "FIXME: Implement getOverlapFraction" << std::endl;
    // std::cout << "FIXME: Implement getOverlapFraction" << std::endl;
    // std::cout << "FIXME: Implement getOverlapFraction" << std::endl;
    return 0.0;
}

void Disk::calculateForces() {
    // kernelCalcDiskForces<<<particle_dim_grid, particle_dim_block>>>(d_positions_x_ptr, d_positions_y_ptr, d_radii_ptr, d_forces_x_ptr, d_forces_y_ptr, d_potential_energy_ptr);
    kernelCalcDiskForces<<<particle_dim_grid, particle_dim_block>>>(positions.x.d_ptr, positions.y.d_ptr, radii.d_ptr, forces.x.d_ptr, forces.y.d_ptr, potential_energy.d_ptr);
}

void Disk::calculateKineticEnergy() {
    // kernelCalculateTranslationalKineticEnergy<<<particle_dim_grid, particle_dim_block>>>(d_velocities_x_ptr, d_velocities_y_ptr, d_masses_ptr, d_kinetic_energy_ptr);
    kernelCalculateTranslationalKineticEnergy<<<particle_dim_grid, particle_dim_block>>>(velocities.x.d_ptr, velocities.y.d_ptr, masses.d_ptr, kinetic_energy.d_ptr);
}

void Disk::calculateForceDistancePairs() {
    force_pairs.resizeAndFill(n_particles * max_neighbors_allocated, 0.0, 0.0);
    distance_pairs.resizeAndFill(n_particles * max_neighbors_allocated, -1.0, -1.0);
    pair_ids.resizeAndFill(n_particles * max_neighbors_allocated, -1L, -1L);
    overlap_pairs.resizeAndFill(n_particles * max_neighbors_allocated, -1.0);
    radsum_pairs.resizeAndFill(n_particles * max_neighbors_allocated, -1.0);

    pos_pairs_i.resizeAndFill(n_particles * max_neighbors_allocated, -1.0, -1.0);
    pos_pairs_j.resizeAndFill(n_particles * max_neighbors_allocated, -1.0, -1.0);
    
    kernelCalcDiskForceDistancePairs<<<particle_dim_grid, particle_dim_block>>>(positions.x.d_ptr, positions.y.d_ptr, force_pairs.x.d_ptr, force_pairs.y.d_ptr, distance_pairs.x.d_ptr, distance_pairs.y.d_ptr, pair_ids.x.d_ptr, pair_ids.y.d_ptr, overlap_pairs.d_ptr, radsum_pairs.d_ptr, radii.d_ptr, static_particle_index.d_ptr, pos_pairs_i.x.d_ptr, pos_pairs_i.y.d_ptr, pos_pairs_j.x.d_ptr, pos_pairs_j.y.d_ptr);
}

void Disk::calculateWallForces() {
    kernelCalcDiskWallForces<<<particle_dim_grid, particle_dim_block>>>(positions.x.d_ptr, positions.y.d_ptr, radii.d_ptr, forces.x.d_ptr, forces.y.d_ptr, potential_energy.d_ptr);
}

void Disk::loadData(const std::string& root) {
    // unify all particle configs
    // add load functionality to configs


    // load config

    // set config

    // load data

    
    // SwapData2D<double> positions = read_2d_swap_data_from_file<double>(last_step_dir + "/positions.dat", particle_config.n_particles, 2);
    // Data1D<double> radii = read_1d_data_from_file<double>(source_path + "system/init/radii.dat", particle_config.n_particles);
}