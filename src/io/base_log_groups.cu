#include "hip/hip_runtime.h"
#include "../../include/io/base_log_groups.h"
#include "../../include/io/orchestrator.h"
#include <string>
#include <iostream>


LogGroupConfigDict config_from_names_lin(std::vector<std::string> log_names, long num_steps, long num_saves, std::string group_name) {
    LogGroupConfigDict config;
    config["log_names"] = log_names;
    config["save_style"] = "lin";
    config["save_freq"] = static_cast<long>(num_steps / num_saves);
    if (config["save_freq"] == 0) {
        config["save_freq"] = 1;
    }
    config["group_name"] = group_name;
    return config;
}

LogGroupConfigDict config_from_names_log(std::vector<std::string> log_names, long num_steps, long num_saves, long min_save_decade, std::string group_name) {
    LogGroupConfigDict config;
    config["log_names"] = log_names;
    config["save_style"] = "log";
    config["reset_save_decade"] = static_cast<long>(num_steps / num_saves);
    if (config["reset_save_decade"] == 0) {
        config["reset_save_decade"] = 1;
    }
    config["min_save_decade"] = min_save_decade;
    config["group_name"] = group_name;
    return config;
}

LogGroupConfigDict config_from_names_lin_everyN(std::vector<std::string> log_names, long save_freq, std::string group_name) {
    LogGroupConfigDict config;
    config["log_names"] = log_names;
    config["save_style"] = "lin";
    config["save_freq"] = save_freq;
    config["group_name"] = group_name;
    return config;
}

LogGroupConfigDict config_from_names(std::vector<std::string> log_names, std::string group_name) {
    LogGroupConfigDict config;
    config["log_names"] = log_names;
    config["group_name"] = group_name;
    return config;
}

BaseLogGroup::BaseLogGroup(LogGroupConfigDict config, Orchestrator& orchestrator) : config(config), orchestrator(orchestrator) {
    log_names = config.get<std::vector<std::string>>("log_names");
    save_style = config.get<std::string>("save_style");
    save_freq = config.get<long>("save_freq");
    reset_save_decade = config.get<long>("reset_save_decade");
    min_save_decade = config.get<long>("min_save_decade");
    multiple = config.get<long>("multiple");
    decade = config.get<long>("decade");
    group_name = config.get<std::string>("group_name");
}

BaseLogGroup::~BaseLogGroup() {
}

void BaseLogGroup::define_dependencies() {
    for (const std::string& log_name : config["log_names"]) {
        if (orchestrator.is_dependent(log_name)) {
            dependencies.insert(log_name);
            has_dependencies = true;
        }
    }
}

void BaseLogGroup::handle_dependencies() {
    for (const std::string& log_name : dependencies) {
        orchestrator.handle_dependencies(log_name);
    }
}

void BaseLogGroup::update_log_status(long step) {
    if (save_style == "lin") {
        should_log = step % save_freq == 0;
    } else if (save_style == "log") {
        if (step > multiple * reset_save_decade) {
            save_freq = min_save_decade;
            multiple += 1;
        }
        if ((step - (multiple - 1) * reset_save_decade) > save_freq * decade) {
            save_freq *= decade;
        }
        if ((step - (multiple - 1) * reset_save_decade) % save_freq == 0) {
            should_log = true;
        } else {
            should_log = false;
        }
    } else {
        std::cout << "ERROR: LogManager::update_log_status: Invalid save style: " << config["save_style"] << std::endl;
        exit(1);
    }
}

ScalarLog::ScalarLog(LogGroupConfigDict config, Orchestrator& orchestrator) : BaseLogGroup(config, orchestrator) {
    unmodified_log_names = get_unmodified_log_names();
}

ScalarLog::~ScalarLog() {
}

void ScalarLog::define_dependencies() {
    for (const std::string& log_name : unmodified_log_names) {
        if (orchestrator.is_dependent(log_name)) {
            dependencies.insert(log_name);
            has_dependencies = true;
        }
    }
}

bool ScalarLog::is_modified(std::string log_name) {
    return log_name.find(modifier) != std::string::npos;
}

void ScalarLog::gather_data(long step) {
    for (size_t i = 0; i < unmodified_log_names.size(); ++i) {
        const auto& name = unmodified_log_names[i];
        double value = orchestrator.get_value<double>(name, step);
        if (is_modified(log_names[i])) {
            std::string mod = get_modifier(log_names[i]);
            value = orchestrator.apply_modifier(mod, value);
        }
        gathered_data[log_names[i]] = value;
    }
}

std::vector<std::string> ScalarLog::get_unmodified_log_names() {
    std::vector<std::string> unmodified_log_names;
    for (auto& log_name : log_names) {
        size_t pos = log_name.find(modifier);
        if (pos != std::string::npos) {
            unmodified_log_names.push_back(log_name.substr(0, pos));
        } else {
            unmodified_log_names.push_back(log_name);
        }
    }
    return unmodified_log_names;
}

std::string ScalarLog::get_modifier(std::string log_name) {
    size_t pos = log_name.find(modifier);
    if (pos != std::string::npos) {
        return log_name.substr(pos + 1);
    } else {
        return "";
    }
}